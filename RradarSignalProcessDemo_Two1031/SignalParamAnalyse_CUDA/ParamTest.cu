#include "hip/hip_runtime.h"
﻿#include "ParamTest.cuh"
#include "CommonKernel.cuh"




void ParamGpuProgressHe(vector<vector<double>>& lep,const vector<complex<double>>& Source_Data)
{
	//int rows = 0;
	//int cols = 0;
	//if (lep.size() > 0)
	//{
	//	cols = lep.size();
	//	if (lep.at(0).size() > 0)
	//	{
	//		rows = lep.at(0).size();
	//	}
	//}
	//else
	//{
	//	return;
	//}
	//CCMat Matrix_Test(256, 1, 1);
	//Matrix_Test[0].x = 
	//float Matrix_Real[][];
	//float Matrix_Imag[][];
	// begin,开始
	const int rows = 128;
	const int cols = 62500;
	const int sizes = rows * cols;

	static float Matrix_vector[sizes];
	static unsigned int NN_Matrix[sizes];
	int aa_cols = 0;
	int bb_rows = 0;
	// 循环赋值
	for (int ii = 0; ii < sizes; ii++)
	{
		//printf("输出一列的每行数据aa_cols:%d,bb_rows:%d\n", aa_cols, bb_rows);//
		Matrix_vector[ii] = lep.at(aa_cols).at(bb_rows);  //先放的是一列的每行数据，再放下一列
		NN_Matrix[ii] = 0;

		if (bb_rows == rows - 1)
		{
			bb_rows = -1;
			if (aa_cols == cols - 1)
			{
				break;
			}
			aa_cols++;
			//continue;
		}
		bb_rows++;
	}
	//printf("Matrix_vector[8000000]:%f\n", Matrix_vector[7999999]);
	////测试总和使用，测试结论正确
	////float total_he_vector(0.0);
	////for (int ii = 0; ii < rows * cols; ii++)
	////{
	////	total_he_vector = total_he_vector + Matrix_vector[ii];
	////}
	////printf("total_he_vector:%f\n", total_he_vector);

	float* d_vector;
	float* d_average;
	hipMalloc((void**)& d_vector, sizes * sizeof(float));
	hipMalloc((void**)& d_average, sizeof(float));
	hipError_t cudaStatus_vector = hipMemcpy(d_vector, Matrix_vector, sizes * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus_vector != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_vector) << std::endl;
	}
	else
	{
		std::cout << "hipMemcpy succeeded." << std::endl;
	}
	hipError_t cudaStatus_daverage = hipMemset(d_average, 0, sizeof(float));
	if (cudaStatus_daverage != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_daverage) << std::endl;
	}
	else
	{
		std::cout << "hipMemcpy succeeded." << std::endl;
	}

	//float* cudaCols;
	//float* cudaRows;
	//hipMalloc((void**)& cudaCols, sizeof(float));
	//hipMemset(cudaCols, cols, sizeof(float));
	//hipMalloc((void**)& cudaRows, sizeof(float));
	//hipMemset(cudaRows, rows, sizeof(float));
	//printf("d_vector:%f\n", d_vector);

	dim3 blockSize(128, 62500);//线程块    //128行，62500列
	int grid_x = (rows + blockSize.x - 1) / blockSize.x;//cols是列，rows是行
	int grid_y = (cols + blockSize.y - 1) / blockSize.y;
	//printf("grid_x:%d,grid_y:%d\n", grid_x, grid_y);
	dim3 gridSize(grid_x, grid_y);//1，1
	//printf("d_vector:%f\n", d_vector);

	hipError_t cudaStatus;//设置cuda核函数状态变量
	//blockSize是并行线程块的数量，blockIdx是一个内置变量;gridSize is 每个线程块对应的线程数，每个线程块的线程被限制在512
	//并行的线程块集合又称为一个线程格(Grid)
	vectorAverage << <blockSize, gridSize >> > (d_vector, rows, cols, d_average);
	cudaStatus = hipGetLastError(); //获取核函数执行后的状态变量状态
	if (cudaStatus != hipSuccess)
	{
		//核函数执行失败
		printf("Kernel execution failed:%s\n", hipGetErrorString(cudaStatus));
	}

	//所有数据的平均值
	float h_average;
	hipError_t cudaStatus_haverage = hipMemcpy(&h_average, d_average, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus_haverage != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_haverage) << '\n'/*std::endl*/;
	}
	else
	{
		std::cout << "hipMemcpy succeeded." << '\n'/*std::endl*/;
	}
	//printf("cudaCols:%f\n", cudaCols);
	printf("h_average:%f\n", h_average);

	h_average /= sizes;
	//第一步利用核函数计算平均值已完成，第二步利用门限值判别超过门限值的所有位置信息
	unsigned int* NN;
	hipError_t cudaMalloc_NN = hipMalloc((void**)& NN, sizes * sizeof(unsigned int));
	if (cudaMalloc_NN != hipSuccess)
	{
		std::cout << "CudaMalloc failed:" << hipGetErrorString(cudaMalloc_NN) << std::endl;
	}
	//printf("sizeof(unsigned int): %d.\n", sizeof(unsigned int));
	hipError_t cudaStatus_NN = hipMemcpy(NN, NN_Matrix, sizes * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus_NN != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_NN) << std::endl;
	}
	//编写一个核函数
	hipError_t cudaStatus_NNfunction;//设置cuda核函数状态变量
	CalculateNN_Array << <blockSize, gridSize >> > (d_vector, rows, cols, h_average * 30, NN);
	cudaStatus_NNfunction = hipGetLastError(); //获取核函数执行后的状态变量状态
	if (cudaStatus_NNfunction != hipSuccess)
	{
		//核函数执行失败
		printf("Kernel execution failed:%s\n", hipGetErrorString(cudaStatus_NNfunction));
	}

	//完成第二步骤将数据与门限判别之后，进行第三步，识别出每一列和行连续是1的索引，并将其输出
	//Source_Data
	//先识别一维原始数据的脉冲起始时间和结束时间，再根据此时间，计算二维带宽






	//printf("sizes:%d\n", sizes);
	//printf("Average: %f\n", h_average);

	hipDeviceSynchronize(); //在此之后，可以确保核函数已经执行完成
	hipFree(d_vector);
	hipFree(d_average);


}



void Test(vector<complex<double>>& temp)
{

	float Matrix_Real[256];
	float Matrix_Imag[256];

	for (int ii = 0; ii < temp.size(); ii++)
	{
		Matrix_Real[ii] = temp.at(ii).real();
		Matrix_Imag[ii] = temp.at(ii).imag();
	}

	const int Mem = 256 * sizeof(float);

	//Gpu申请内存
	float* matrix_cuda_real;
	float* matrix_cuda_imag;

	hipMalloc((void**)& matrix_cuda_real, Mem);
	hipMalloc((void**)& matrix_cuda_imag, Mem);

	//将cpu数据拷至gpu上
	hipMemcpy(matrix_cuda_real, Matrix_Real, Mem, hipMemcpyHostToDevice);
	hipMemcpy(matrix_cuda_imag, Matrix_Imag, Mem, hipMemcpyHostToDevice);

	int threadsPerBlock1 = 256;  //每个线程块的线程数为256；
	int blocksPerGrid1 = (256 + threadsPerBlock1 - 1) / threadsPerBlock1;

	//printf("%d,%d\n", blocksPerGrid1, threadsPerBlock1);
	CCMat Matrix_Test(256, 1, 1);
	//hipStream_t Stream;
	//ComplexMat << <blocksPerGrid1, threadsPerBlock1 >> > (Matrix_Test.FirstAddr(), matrix_cuda_real, matrix_cuda_imag, 256);
	tools::SaveDeviceComplexMatrix(Matrix_Test.FirstAddr(), Matrix_Test.Row(), Matrix_Test.Col(), "save\\gpuMatrix.txt");

	//const dim3 gridSize(2);
	//const dim3 blockSize(3);
	//printf("start\n");
	//hello_from_gpu << <gridSize, blockSize >> > ();
	//printf("endn\n");
	//hipDeviceSynchronize();

}
//GPU计算的应用前景再很大程度上取决于能否从许多问题中发掘出大规模并行性;
__global__ void vectorAverage(float* d_vector, int width, int height, float* d_average)
{                                              //width is 128; height is 62500;
	                                         
	int idx = blockIdx.x * blockDim.x + threadIdx.x;   //blockDim is 线程块中每一维的线程数量
	//线程块 128 * 62500二维;blockIdx.x是从0~127，blockDim.x是每一个块内的线程数 is 1，threadIdx.x是0;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	//线程块 128 * 62500二维;blockIdx.y是从0~62499，blockDim.y is 1，threadIdx.y是0;
	//printf("%d,%d,%d,%d.\n", blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
	//printf("%d,%d.\n", blockIdx.x, blockIdx.y);
	if (idx < width && idy < height)
	{
		int index = idy * width + idx;
		//printf("%d.\n", index);
		atomicAdd(d_average, d_vector[index]);
	}
}

__global__ void CalculateNN_Array(float* d_vector, int width, int heigth, float h_average, unsigned int* NN)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < width && idy < heigth)
	{
		int index = idy * width + idx;
		if (d_vector[index] > h_average)
		{
			NN[index] = 1;
		}
	}
}

//__global__ void ComplexMat(hipComplex* Res, float* Real, float* Imag, int elements)
//{
//	// 线程块中线程量 * 线程块x + 线程x
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (i < elements)
//	{
//		Res[i].x = Real[i];
//		Res[i].x = Imag[i];
//	}
//	//return __global__ void();
//}

//int main()
//{
//	const dim3 gridSize(2);
//	const dim3 blockSize(3);
//	printf("start\n");
//	hello_from_gpu << <gridSize, blockSize >> > ();
//	printf("endn\n");
//	hipDeviceSynchronize();
//	return 0;
//}