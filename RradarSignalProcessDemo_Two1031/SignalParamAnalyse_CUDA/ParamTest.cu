#include "hip/hip_runtime.h"
﻿#include "ParamTest.cuh"
#include <numeric>
#include "CommonKernel.cuh"




void ParamGpuProgressHe(vector<vector<double>>& lep,const vector<complex<double>>& Source_Data)
{
	//int rows = 0;
	//int cols = 0;
	//if (lep.size() > 0)
	//{
	//	cols = lep.size();
	//	if (lep.at(0).size() > 0)
	//	{
	//		rows = lep.at(0).size();
	//	}
	//}
	//else
	//{
	//	return;
	//}
	//CCMat Matrix_Test(256, 1, 1);
	//Matrix_Test[0].x = 
	//float Matrix_Real[][];
	//float Matrix_Imag[][];
	// begin,开始
	const int rows = 128;
	const int cols = 62500;
	const int sizes = rows * cols;

	static float Matrix_vector[sizes];
	static unsigned int NN_Matrix[sizes];
	int aa_cols = 0;
	int bb_rows = 0;
	// 循环赋值
	for (int ii = 0; ii < sizes; ii++)
	{
		//printf("输出一列的每行数据aa_cols:%d,bb_rows:%d\n", aa_cols, bb_rows);//
		Matrix_vector[ii] = lep.at(aa_cols).at(bb_rows);  //先放的是一列的每行数据，再放下一列
		NN_Matrix[ii] = 0;

		if (bb_rows == rows - 1)
		{
			bb_rows = -1;
			if (aa_cols == cols - 1)
			{
				break;
			}
			aa_cols++;
			//continue;
		}
		bb_rows++;
	}
	//printf("Matrix_vector[8000000]:%f\n", Matrix_vector[7999999]);
	////测试总和使用，测试结论正确
	////float total_he_vector(0.0);
	////for (int ii = 0; ii < rows * cols; ii++)
	////{
	////	total_he_vector = total_he_vector + Matrix_vector[ii];
	////}
	////printf("total_he_vector:%f\n", total_he_vector);

	float* d_vector;
	float* d_average;
	hipMalloc((void**)& d_vector, sizes * sizeof(float));
	hipMalloc((void**)& d_average, sizeof(float));
	hipError_t cudaStatus_vector = hipMemcpy(d_vector, Matrix_vector, sizes * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus_vector != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_vector) << std::endl;
	}
	else
	{
		std::cout << "hipMemcpy succeeded." << std::endl;
	}
	hipError_t cudaStatus_daverage = hipMemset(d_average, 0, sizeof(float));
	if (cudaStatus_daverage != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_daverage) << std::endl;
	}
	else
	{
		std::cout << "hipMemcpy succeeded." << std::endl;
	}

	//float* cudaCols;
	//float* cudaRows;
	//hipMalloc((void**)& cudaCols, sizeof(float));
	//hipMemset(cudaCols, cols, sizeof(float));
	//hipMalloc((void**)& cudaRows, sizeof(float));
	//hipMemset(cudaRows, rows, sizeof(float));
	//printf("d_vector:%f\n", d_vector);

	dim3 blockSize(128, 62500);//线程块    //128行，62500列
	int grid_x = (rows + blockSize.x - 1) / blockSize.x;//cols是列，rows是行
	int grid_y = (cols + blockSize.y - 1) / blockSize.y;
	//printf("grid_x:%d,grid_y:%d\n", grid_x, grid_y);
	dim3 gridSize(grid_x, grid_y);//1，1
	//printf("d_vector:%f\n", d_vector);

	hipError_t cudaStatus;//设置cuda核函数状态变量
	//blockSize是并行线程块的数量，blockIdx是一个内置变量;gridSize is 每个线程块对应的线程数，每个线程块的线程被限制在512
	//并行的线程块集合又称为一个线程格(Grid)
	vectorAverage << <blockSize, gridSize >> > (d_vector, rows, cols, d_average);
	cudaStatus = hipGetLastError(); //获取核函数执行后的状态变量状态
	if (cudaStatus != hipSuccess)
	{
		//核函数执行失败
		printf("Kernel execution failed:%s\n", hipGetErrorString(cudaStatus));
	}

	//所有数据的平均值
	float h_average;
	hipError_t cudaStatus_haverage = hipMemcpy(&h_average, d_average, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus_haverage != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_haverage) << '\n'/*std::endl*/;
	}
	else
	{
		std::cout << "hipMemcpy succeeded." << '\n'/*std::endl*/;
	}
	//printf("cudaCols:%f\n", cudaCols);
	printf("h_average:%f\n", h_average);

	h_average /= sizes;
	//第一步利用核函数计算平均值已完成，第二步利用门限值判别超过门限值的所有位置信息
	unsigned int* NN;
	hipError_t cudaMalloc_NN = hipMalloc((void**)& NN, sizes * sizeof(unsigned int));
	if (cudaMalloc_NN != hipSuccess)
	{
		std::cout << "CudaMalloc failed:" << hipGetErrorString(cudaMalloc_NN) << std::endl;
	}
	//printf("sizeof(unsigned int): %d.\n", sizeof(unsigned int));
	hipError_t cudaStatus_NN = hipMemcpy(NN, NN_Matrix, sizes * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus_NN != hipSuccess)
	{
		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_NN) << std::endl;
	}
	//编写一个核函数
	hipError_t cudaStatus_NNfunction;//设置cuda核函数状态变量
	CalculateNN_Array << <blockSize, gridSize >> > (d_vector, rows, cols, h_average * 30, NN);
	cudaStatus_NNfunction = hipGetLastError(); //获取核函数执行后的状态变量状态
	if (cudaStatus_NNfunction != hipSuccess)
	{
		//核函数执行失败
		printf("Kernel execution failed:%s\n", hipGetErrorString(cudaStatus_NNfunction));
	}

	//完成第二步骤将数据与门限判别之后，进行第三步，识别出每一列和行连续是1的索引，并将其输出
	//Source_Data
	//先识别一维原始数据的脉冲起始时间和结束时间，再根据此时间，计算二维带宽

	////根据时域数据计算脉冲上升沿下降沿时刻
	//vector<int> UpVec;           //上升沿vector
	//vector<int> DownVec;         //下降沿vector
	//vector<int> PulseWidthVec;   //对应脉宽
	//double CurrentTime = 0.0;
	//double m_JamInitSampleFs = 0.0;
	////m_pSignalProcessor->GetJamInitSampleFs(m_JamInitSampleFs);
	//// 参数估计--估计脉冲上升沿、下降沿、脉宽
	//if (Source_Data.size() > 0)
	//{
	//	ParameterAnalysis(Source_Data, UpVec, DownVec, PulseWidthVec, CurrentTime, m_JamInitSampleFs);//输出雷达信号参数单位(毫秒)[上升沿、下降沿、脉宽]
	//}



	//for (int ii = 0;ii<UpVec.size();ii++)
	//{
	//	int begin_data = UpVec.at(ii);  //起始位置
	//	int end_data = DownVec.at(ii);  //结束位置

	//	int ThreadNum = end_data - begin_data;

	//	int* Array;
	//	hipError_t cudaMalloc_Array = hipMalloc((void**)& Array, ThreadNum * 2 * sizeof(int));
	//	if (cudaMalloc_Array != hipSuccess)
	//	{
	//		std::cout << "CudaMalloc failed:" << hipGetErrorString(cudaMalloc_Array) << std::endl;
	//	}
	//	hipError_t cudaStatus_Array = hipMemcpy(Array, 0, ThreadNum * 2 * sizeof(int), hipMemcpyHostToDevice);
	//	if (cudaStatus_Array != hipSuccess)
	//	{
	//		std::cout << "CudaMemcpy failed:" << hipGetErrorString(cudaStatus_Array) << std::endl;
	//	}


	//}




	//printf("sizes:%d\n", sizes);
	//printf("Average: %f\n", h_average);

	hipDeviceSynchronize(); //在此之后，可以确保核函数已经执行完成
	hipFree(d_vector);
	hipFree(d_average);


}

void ParameterAnalysis(const vector<complex<double>>& AllRadarData, vector<int>& UpData, vector<int>& DownData, vector<int>& PulseWidth, double& time, const double Fs)
{
	// 估计AllRadarData的上升沿、下降沿和脉宽，并存入对应的vector中，便于后续进行分选处理
	int PulseUpFlag = 0;
	int PulseDownFlag = 0;
	int PulseUpPos = 0;
	int PulseDownPos = 0;

	// 1、计算abs值
	vector<double> HeDataABS;
	HeDataABS.resize(AllRadarData.size());
	for (int ii = 0; ii < AllRadarData.size(); ii++)
	{
		float RealData = AllRadarData.at(ii).real();
		float ImagData = AllRadarData.at(ii).imag();
		HeDataABS[ii] = sqrt(RealData * RealData + ImagData * ImagData);
	}

	// 2、利用前面纯噪声部分计算噪声功率和方差
	vector<double> DataRead;
	DataRead.resize(HeDataABS.size());
	for (int jj = 0; jj < HeDataABS.size(); jj++)
	{
		DataRead[jj] = HeDataABS[jj] * HeDataABS[jj];   //计算功率
	}
	HeDataABS.clear();

	// 检测所用参数
	int PulseFlag = 0;
	int	CntBegin = 0;
	int CntEnd = 0;
	double NoisePower = 0.0;
	double DataPower = 0.0;
	int BeginFrameThrh = 128;
	int EndFrameThrh = 128;
	int FrameSize = 128; //帧长度
	NoisePower = accumulate(DataRead.begin(), DataRead.begin() + FrameSize, 0.0) / FrameSize; //计算前128个点的噪声功率均值

	vector<double> data_buff;
	data_buff.resize(FrameSize);
	// 3、能量检测方法检测脉冲边沿
	vector <int> UpPoint;
	vector <int> DownPoint;
	vector <int> PulseWidthPoint;
	for (int k = 0; k < DataRead.size(); k++)
	{

		for (int j = 1; j < FrameSize; j++)
		{
			data_buff[FrameSize - j] = data_buff[FrameSize - j - 1];
		}
		data_buff[0] = DataRead[k];

		//if (k == 800)
		//{
		//	int oo = 0;  //测试
		//}
		DataPower = accumulate(data_buff.begin(), data_buff.end(), 0.0) / data_buff.size();  //计算该帧信号平均功率

		if (DataPower > 1.5 * NoisePower)
		{
			CntEnd = 0;
			CntBegin = CntBegin + 1;

			if (CntBegin >= BeginFrameThrh) // 连续有超过BeginFrameThrh帧信号超过检测门限，则认为是脉冲开始
			{
				if (PulseFlag == 0) // 此前还未检测到脉冲
				{
					// 寻找精确的脉冲起始位置
					PulseUpPos = k - FrameSize;
					UpPoint.push_back(PulseUpPos);
				}

				PulseUpFlag = 1;
				PulseFlag = 1;

			}
		}
		else
		{
			if (PulseFlag == 1)
			{
				CntEnd = CntEnd + 1;
				if (CntEnd >= EndFrameThrh)  //有连续超过EndFrameThrh帧信号低于检测门限，则认为是脉冲结束
				{
					// 寻找精确的脉冲结束位置
					PulseDownPos = k - FrameSize - 127;
					DownPoint.push_back(PulseDownPos);
					PulseWidthPoint.push_back(PulseDownPos - PulseUpPos);
					PulseDownFlag = 1;
					PulseFlag = 0;
					CntBegin = 0;
				}
			}
			else
			{
				CntBegin = 0;
				CntEnd = 0;
			}
		}
	}

	// 4、点数换算时间--将点数Pos位置换算到时间上(单位/毫秒)
	for (int aa = 0; aa < UpPoint.size(); aa++)
	{
		//double Tmp_UpTime;
		//Tmp_UpTime = ((time + UpPoint[aa] / Fs) * 1e3);
		UpData.push_back(UpPoint.at(aa));
	}
	for (int bb = 0; bb < DownPoint.size(); bb++)
	{
		//double Tmp_DownTime;
		//Tmp_DownTime = ((time + DownPoint[bb] / Fs) * 1e3);
		DownData.push_back(DownPoint.at(bb));
	}
	for (int cc = 0; cc < PulseWidthPoint.size(); cc++)
	{
		//double Tmp_PulseWidthTime;
		//Tmp_PulseWidthTime = ((time * 1e3 + PulseWidthPoint[cc] / Fs) * 1e3);
		PulseWidth.push_back(PulseWidthPoint.at(cc));
	}



	//// 4、点数换算时间--将点数Pos位置换算到时间上(单位/毫秒)
	//for (int aa = 0; aa < UpPoint.size(); aa++)
	//{
	//	double Tmp_UpTime;
	//	Tmp_UpTime = ((time + UpPoint[aa] / Fs) * 1e3);
	//	UpData.push_back(Tmp_UpTime);
	//}
	//for (int bb = 0; bb < DownPoint.size(); bb++)
	//{
	//	double Tmp_DownTime;
	//	Tmp_DownTime = ((time + DownPoint[bb] / Fs) * 1e3);
	//	DownData.push_back(Tmp_DownTime);
	//}
	//for (int cc = 0; cc < PulseWidthPoint.size(); cc++)
	//{
	//	double Tmp_PulseWidthTime;
	//	Tmp_PulseWidthTime = ((time * 1e3 + PulseWidthPoint[cc] / Fs) * 1e3);
	//	PulseWidth.push_back(Tmp_PulseWidthTime);
	//}
}


void Test(vector<complex<double>>& temp)
{

	float Matrix_Real[256];
	float Matrix_Imag[256];

	for (int ii = 0; ii < temp.size(); ii++)
	{
		Matrix_Real[ii] = temp.at(ii).real();
		Matrix_Imag[ii] = temp.at(ii).imag();
	}

	const int Mem = 256 * sizeof(float);

	//Gpu申请内存
	float* matrix_cuda_real;
	float* matrix_cuda_imag;

	hipMalloc((void**)& matrix_cuda_real, Mem);
	hipMalloc((void**)& matrix_cuda_imag, Mem);

	//将cpu数据拷至gpu上
	hipMemcpy(matrix_cuda_real, Matrix_Real, Mem, hipMemcpyHostToDevice);
	hipMemcpy(matrix_cuda_imag, Matrix_Imag, Mem, hipMemcpyHostToDevice);

	int threadsPerBlock1 = 256;  //每个线程块的线程数为256；
	int blocksPerGrid1 = (256 + threadsPerBlock1 - 1) / threadsPerBlock1;

	//printf("%d,%d\n", blocksPerGrid1, threadsPerBlock1);
	CCMat Matrix_Test(256, 1, 1);
	//hipStream_t Stream;
	//ComplexMat << <blocksPerGrid1, threadsPerBlock1 >> > (Matrix_Test.FirstAddr(), matrix_cuda_real, matrix_cuda_imag, 256);
	tools::SaveDeviceComplexMatrix(Matrix_Test.FirstAddr(), Matrix_Test.Row(), Matrix_Test.Col(), "save\\gpuMatrix.txt");

	//const dim3 gridSize(2);
	//const dim3 blockSize(3);
	//printf("start\n");
	//hello_from_gpu << <gridSize, blockSize >> > ();
	//printf("endn\n");
	//hipDeviceSynchronize();

}
//GPU计算的应用前景再很大程度上取决于能否从许多问题中发掘出大规模并行性;
__global__ void vectorAverage(float* d_vector, int width, int height, float* d_average)
{                                              //width is 128; height is 62500;
	                                         
	int idx = blockIdx.x * blockDim.x + threadIdx.x;   //blockDim is 线程块中每一维的线程数量
	//线程块 128 * 62500二维;blockIdx.x是从0~127，blockDim.x是每一个块内的线程数 is 1，threadIdx.x是0;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	//线程块 128 * 62500二维;blockIdx.y是从0~62499，blockDim.y is 1，threadIdx.y是0;
	//printf("%d,%d,%d,%d.\n", blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
	//printf("%d,%d.\n", blockIdx.x, blockIdx.y);
	if (idx < width && idy < height)
	{
		int index = idy * width + idx;
		//printf("%d.\n", index);
		atomicAdd(d_average, d_vector[index]);
	}
}

__global__ void CalculateNN_Array(float* d_vector, int width, int heigth, float h_average, unsigned int* NN)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < width && idy < heigth)
	{
		int index = idy * width + idx;
		if (d_vector[index] > h_average)
		{
			NN[index] = 1;
		}
	}
}

//__global__ void ComplexMat(hipComplex* Res, float* Real, float* Imag, int elements)
//{
//	// 线程块中线程量 * 线程块x + 线程x
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (i < elements)
//	{
//		Res[i].x = Real[i];
//		Res[i].x = Imag[i];
//	}
//	//return __global__ void();
//}

//int main()
//{
//	const dim3 gridSize(2);
//	const dim3 blockSize(3);
//	printf("start\n");
//	hello_from_gpu << <gridSize, blockSize >> > ();
//	printf("endn\n");
//	hipDeviceSynchronize();
//	return 0;
//}