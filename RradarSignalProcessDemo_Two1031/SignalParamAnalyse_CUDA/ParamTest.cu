#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <hip/hip_vector_types.h>

__global__ void hello_from_gpu()
{
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	printf("%d,%d.\n", bid, tid);
}

int main()
{
	const dim3 gridSize(2);
	const dim3 blockSize(3);
	printf("start\n");
	hello_from_gpu << <gridSize, blockSize >> > ();
	printf("endn\n");
	hipDeviceSynchronize();
	return 0;
}