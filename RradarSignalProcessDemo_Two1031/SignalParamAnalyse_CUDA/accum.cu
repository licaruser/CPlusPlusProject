#include "hip/hip_runtime.h"
#include "accum.cuh"

__global__ void ReducePartSumKernel(float* input, float* part_sum,
	unsigned int SumBlockNum, unsigned int Len,
	unsigned int part_num)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < SumBlockNum)
	{
		for (int i = 0; i < part_num; i++)
		{
			part_sum[idx] = part_sum[idx] + input[idx * part_num];
		}
	}
}

__global__ void ScanWithBaseSum(float* data, float* part_sum,float* output,
	unsigned int SumBlockNum, unsigned int Len,
	unsigned int part_num) 
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Len)
	{
		unsigned SumBlockBias = idx / part_num;
		unsigned XBias = idx % part_num;
		
		output[idx] = 0.0;

		__syncthreads();

		// ��������͵���
		for (int i = 0; i < SumBlockBias; i++)
		{
			output[idx] = output[idx] + part_sum[i];
		}

		// ���������
		for (int i = 1; i < XBias; i++)
		{
			output[idx] = output[idx] + data[SumBlockBias * part_num + i];
		}

	}

}

// ǰ׺��
FCMat accum(FCMat& Data, hipStream_t stream)
{
	auto XLen = Data.elements();

	int part_num = 1024;
	unsigned int SumBlockNum = XLen / part_num + 1;
	int threadsPerBlock = 1024;
	int blocksPerGrid = (SumBlockNum + threadsPerBlock - 1) / threadsPerBlock;

	// �����part���ۼӺ�
	FCMat SumTemp(SumBlockNum, 1, 1);
	ReducePartSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data.FirstAddr(), SumTemp.FirstAddr(),SumBlockNum, XLen, part_num);

	FCMat Result(Data.dims(0), Data.dims(1), Data.dims(2));
	blocksPerGrid = (XLen + threadsPerBlock - 1) / threadsPerBlock;
	ScanWithBaseSum << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data.FirstAddr(), SumTemp.FirstAddr(), Result.FirstAddr(),
		SumBlockNum, XLen, part_num);

	SumTemp.Free();

	return Result;
}

// ǰ׺��
void accum(FCMat& Data, FCMat& Result, hipStream_t stream)
{
	Result.Resize(Data.dims(0), Data.dims(1), Data.dims(2));

	auto XLen = Data.elements();

	int part_num = 1024;
	unsigned int SumBlockNum = XLen / part_num + 1;
	int threadsPerBlock = 1024;
	int blocksPerGrid = (SumBlockNum + threadsPerBlock - 1) / threadsPerBlock;

	// �����part���ۼӺ�
	FCMat SumTemp(SumBlockNum, 1, 1);
	ReducePartSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data.FirstAddr(), SumTemp.FirstAddr(), SumBlockNum, XLen, part_num);

	
	blocksPerGrid = (XLen + threadsPerBlock - 1) / threadsPerBlock;
	ScanWithBaseSum << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data.FirstAddr(), SumTemp.FirstAddr(), Result.FirstAddr(),
		SumBlockNum, XLen, part_num);

	SumTemp.Free();

}

