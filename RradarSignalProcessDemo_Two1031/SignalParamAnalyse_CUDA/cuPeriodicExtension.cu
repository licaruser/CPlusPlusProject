#include "hip/hip_runtime.h"
#include "cuPeriodicExtension.cuh"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""





__global__ void cuPeriodicExtensionKernel(hipComplex* Data, hipComplex* Result, int Period, int Th, int DataLen, int ResultLen)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < ResultLen)
	{
		unsigned int index = idx % DataLen;
		unsigned int index2 = idx % Period;

		if (index2 > Th)
		{
			Result[idx] = Data[index];
		}
		else
		{
			Result[idx].x = 0.0;
			Result[idx].y = 0.0;
		}
	}
}

/*
��������
data: �������һ������
result:�������غ������
Period:ѭ������������Ӧ�շ����ڵĵ���
Th:������Ч������Ӧ����ʱ��ĵ���

example:
Period: 4
Th: 2
data:		1 2 3 4 5 6 1 2 3 4 5 6
��������:	1 1 0 0 1 1 0 0 1 1 0 0 
result:		0 0 3 4 0 0 1 2 0 0 5 6 
*/
void PeriodicExtension(CCMat& Data, CCMat& Result, int Period, int Th, hipStream_t Stream)
{

	auto DataLen = Data.elements();
	auto ResultLen = Result.elements();


	//// �����߳̾���ָ��GPU�̷߳���
	int threadsPerBlock = 1024;
	unsigned int blocksPerGrid = (ResultLen + threadsPerBlock - 1) / threadsPerBlock;

	cuPeriodicExtensionKernel << < blocksPerGrid, threadsPerBlock, 0, Stream >> >(Data.FirstAddr(), Result.FirstAddr(), Period, Th, DataLen, ResultLen);

}




