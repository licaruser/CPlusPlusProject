#include "hip/hip_runtime.h"
#include "CommonKernel.cuh"

__global__ void hello_from_gpu()
{

	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	printf("%d,%d.\n", bid, tid);

}

/*������ʱ����������*/
__global__ void TBaseGen(double *t, double fs, double t0, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		t[i] = t0 + i / fs;
	}
}

/*�������Ӻ˺���
������*c = *a + *b
*/
__global__ void ComplexAddKernel(const hipComplex *a, const hipComplex *b, hipComplex *c, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		c[i].x = a[i].x + b[i].x;
		c[i].y = a[i].y + b[i].y;
	}
}


/*���������˺���
������*b = *a
*/
__global__ void ComplexCopyKernel(const hipComplex *a, hipComplex *b, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		b[i].x = a[i].x;
		b[i].y = a[i].y;
	}
}

/*dB
������*b = 20*log10(*a)
*/
__global__ void dBKernel(const float *a, float* b, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		b[i] = 20 * log10(a[i]);
	}
}

/*idB
������*b = pow(10, *a / 20)
*/
__global__ void idBKernel(const float *a, float* b, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		b[i] = pow(10.0, a[i] / 20.0);
	}
}

/*��ʵ������תΪ�鲿Ϊ0�ĸ�������
������*a = *(a+0i)
*/
__global__ void ComplexKernel(const float *a, hipComplex* b, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		b[i].x = a[i];
		b[i].y = 0.0;
	}

}


/*����
������*a = *(a+0i)
*/
__global__ void ConjKernel(hipComplex *data, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		data[i].x = data[i].x;
		data[i].y = -data[i].y;
	}

}
/*������ʵ��ƴ��һ������
������*Res = *(Real+Imagi)
*/
__global__ void ComplexMat(hipComplex *Res, float *Real, float *Imag, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i].x = Real[i];
		Res[i].y = Imag[i];
	}
}


//����ת��
__global__ void TransposeKernel(hipComplex* in, hipComplex* out, int Rows, int Cols, int Bands)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		out[B_bias * (Rows * Cols) + R_bias * Cols + C_bias] = in[idx];

	}
}
__global__ void TransposeKernel(float* in, float* out, int Rows, int Cols, int Bands)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		out[B_bias * (Rows * Cols) + R_bias * Cols + C_bias] = in[idx];

	}
}
__global__ void TransposeKernel(int* in, int* out, int Rows, int Cols, int Bands)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		out[B_bias * (Rows * Cols) + R_bias * Cols + C_bias] = in[idx];

	}
}


//����ת��
__global__ void CTransposeKernel(hipComplex* in, hipComplex* out, int Rows, int Cols, int Bands)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		out[B_bias * (Rows * Cols) + R_bias * Cols + C_bias].x = in[idx].x;
		out[B_bias * (Rows * Cols) + R_bias * Cols + C_bias].y = -in[idx].y;

	}
}


/*ȡʵ��
������*Real = real(*data��
*/
__global__ void Realkernel(hipComplex *Data, float *Real, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Real[i] = Data[i].x;
	}
}

/*ȡ�鲿
������*Imag = imag(*data��
*/
__global__ void Imagkernel(hipComplex *Data, float *Imag, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Imag[i] = Data[i].y;
	}
}

/*��CCMat��abs
*/
__global__ void Abskernel(hipComplex *DataIn, float *DataOut, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		DataOut[i] = sqrt(DataIn[i].x * DataIn[i].x + DataIn[i].y * DataIn[i].y);
	}
}


// ���Ե�ͨ�ź���
__global__ void LowPass(hipComplex *Res, int StartPoint, int EndPoint, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		if (i >= StartPoint && i < EndPoint)
		{
			Res[i].x = 0;
			Res[i].y = 0;
		}
	}
}

/*��������������
������*Res = *Data1 .* *Data2
*/
__global__ void DotMulKernal(hipComplex *Data1, hipComplex *Data2, hipComplex *Res, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i] = hipCmulf(Data1[i], Data2[i]);
	}
}

__global__ void DotMul2Kernal(hipComplex *Data1, hipComplex *Data2, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Data1[i] = hipCmulf(Data1[i], Data2[i]);
	}
}

/*���������������
������*Res = *Data1 + *Data2
*/
__global__ void MatAddKernal(hipComplex *Data1, hipComplex *Data2, hipComplex *Res, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i].x = Data1[i].x + Data2[i].x;
		Res[i].y = Data1[i].y + Data2[i].y;
	}
}

/*���������������
������*Res = *Data1 + *Data2 + *Data3
*/
__global__ void MatAddKernal(hipComplex *Data1, hipComplex *Data2, hipComplex* Data3, hipComplex *Res, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i].x = Data1[i].x + Data2[i].x + Data3[i].x;
		Res[i].y = Data1[i].y + Data2[i].y + Data3[i].y;
	}
}

__global__ void MatAddComplexKernal(hipComplex *Data1, hipComplex Data2, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Data1[i].x = Data1[i].x + Data2.x;
		Data1[i].y = Data1[i].y + Data2.y;
	}
}

/*����������ʵ������
������*Data1.x = *Data1.x * *Data2
*Data1.y = *Data1.y * *Data2
*/
__global__ void MatMulKernal(hipComplex *Data1, float *Data2, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Data1[i].x = Data1[i].x * Data2[i];
		Data1[i].y = Data1[i].y * Data2[i];
	}
}


/*���������float
������*Res = *Data1 / Data2
*/
__global__ void C2FDivKernal(hipComplex *Data1, float Data2, hipComplex *Res, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i].x = Data1[i].x / Data2;
		Res[i].y = Data1[i].y / Data2;
	}
}

__global__ void C2FDiv2Kernal(hipComplex *Data1, float Data2,int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Data1[i].x = Data1[i].x / Data2;
		Data1[i].y = Data1[i].y / Data2;
	}
}

/*���������float
������*Res = *Data1 * Data2
*/
__global__ void C2FMulKernal(hipComplex *Data1, float Data2, hipComplex *Res, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i].x = Data1[i].x * Data2;
		Res[i].y = Data1[i].y * Data2;
	}
}

/*float�����float
������*Res = *Data1 * Data2
*/
__global__ void F2FMulKernal(float *Data1, float Data2, float *Res, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Res[i] = Data1[i] * Data2;
	}
}

/*��ȡ���
��������Data
*/
__global__ void SliceKernel(hipComplex *Data, hipComplex *Result, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		if ((R_bias >= RStart && R_bias <= REnd) &&
			(C_bias >= CStart && C_bias <= CEnd) &&
			(B_bias >= BStart && B_bias <= BEnd))
		{
			unsigned int ResultRowNum = (REnd - RStart + 1);
			unsigned int ResultColNum = (CEnd - CStart + 1);
			Result[(B_bias - BStart) * ResultRowNum * ResultColNum + (C_bias - CStart) * ResultRowNum + (R_bias - RStart)] = Data[idx];

		}

	}


}
__global__ void SliceKernel(float *Data, float *Result, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		if ((R_bias >= RStart && R_bias <= REnd) &&
			(C_bias >= CStart && C_bias <= CEnd) &&
			(B_bias >= BStart && B_bias <= BEnd))
		{
			unsigned int ResultRowNum = (REnd - RStart + 1);
			unsigned int ResultColNum = (CEnd - CStart + 1);
			Result[(B_bias - BStart) * ResultRowNum * ResultColNum + (C_bias - CStart) * ResultRowNum + (R_bias - RStart)] = Data[idx];

		}

	}


}
__global__ void SliceKernel(int *Data, int *Result, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows*Cols*Bands)
	{
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		if ((R_bias >= RStart && R_bias <= REnd) &&
			(C_bias >= CStart && C_bias <= CEnd) &&
			(B_bias >= BStart && B_bias <= BEnd))
		{
			unsigned int ResultRowNum = (REnd - RStart + 1);
			unsigned int ResultColNum = (CEnd - CStart + 1);
			Result[(B_bias - BStart) * ResultRowNum * ResultColNum + (C_bias - CStart) * ResultRowNum + (R_bias - RStart)] = Data[idx];

		}

	}


}

/*��������չ
��������Data��չΪResult���ȣ�������չ��
Data : (Rows, Cols, Bands)
Result : (Rows + abs(AddLength), Cols, Bands)
AddLength : ��չ�������Ҹ���
Value : ��չ��ֵ
*/
__global__ void ExtendKernel(float* Data, float* Result, int AddLength, float Value, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	int ResultRows = Rows + abs(AddLength);
	if (idx < ResultRows * Cols * Bands)
	{
		/*resultƫ��*/
		// ������
		unsigned int B_bias = idx / (ResultRows * Cols);
		// ������
		unsigned int C_bias = (idx % (ResultRows * Cols)) / ResultRows;
		// ������
		unsigned int R_bias = (idx % (ResultRows * Cols)) % ResultRows;

		// ����չ
		if (AddLength > 0)
		{
			// ��չ����
			if (R_bias >= Rows)
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Value;
			}
			// ����չ����
			else
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Data[B_bias * Rows * Cols + C_bias * Rows + R_bias];
			}
		}
		// ����չ
		else
		{
			// ��չ����
			if (R_bias < -AddLength)
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Value;
			}
			// ����չ����
			else
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Data[B_bias * Rows * Cols + C_bias * Rows + R_bias + AddLength];
			}
		}
	}
}


/*��������չ
��������Data��չΪResult���ȣ�������չ��
Data : (Rows, Cols, Bands)
Result : (Rows + abs(AddLength), Cols, Bands)
AddLength : ��չ�������Ҹ���
Value : ��չ��ֵ
*/
__global__ void ExtendKernel(hipComplex* Data, hipComplex* Result, int AddLength, hipComplex Value, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	int ResultRows = Rows + abs(AddLength);
	if (idx < ResultRows * Cols * Bands)
	{
		/*resultƫ��*/
		// ������
		unsigned int B_bias = idx / (ResultRows * Cols);
		// ������
		unsigned int C_bias = (idx % (ResultRows * Cols)) / ResultRows;
		// ������
		unsigned int R_bias = (idx % (ResultRows * Cols)) % ResultRows;

		// ����չ
		if (AddLength > 0)
		{
			// ��չ����
			if (R_bias >= Rows)
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Value;
			}
			// ����չ����
			else
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Data[B_bias * Rows * Cols + C_bias * Rows + R_bias];
			}
		}
		// ����չ
		else
		{
			// ��չ����
			if (R_bias < -AddLength)
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Value;
			}
			// ����չ����
			else
			{
				Result[B_bias * ResultRows * Cols + C_bias * ResultRows + R_bias] = Data[B_bias * Rows * Cols + C_bias * Rows + R_bias + AddLength];
			}
		}
	}
}

/*��ֵɸѡ
������
Data		: (Rows, Cols, Bands)
Result		: (Rows + abs(AddLength), Cols, Bands)
Th			: ��ֵ
CompareFlag	: �Ƚ�����{'<','>','<=','>=','=='}
����
Th = 3; CompareFlag = '>'
Data	:  4 5 6 1 8 3 2 4
Result	:  1 1 1 0 1 0 0 1
*/
__global__ void CompareKernel(float* Data, bool* Result, float Th, char CompareFlag, unsigned int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		if (CompareFlag == '<')
		{
			if (Data[idx] < Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '>')
		{
			if (Data[idx] > Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '<=')
		{
			if (Data[idx] <= Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '>=')
		{
			if (Data[idx] >= Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '==')
		{
			if (Data[idx] == Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
	}
}
__global__ void CompareKernel(hipComplex* Data, bool* Result, float Th, char CompareFlag, unsigned int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		if (CompareFlag == '<')
		{
			if (hipCabsf(Data[idx]) < Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '>')
		{
			if (hipCabsf(Data[idx]) > Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '<=')
		{
			if (hipCabsf(Data[idx]) <= Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '>=')
		{
			if (hipCabsf(Data[idx]) >= Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
		else if (CompareFlag == '==')
		{
			if (hipCabsf(Data[idx]) == Th)
			{
				Result[idx] = true;
			}
			else
			{
				Result[idx] = false;
			}
		}
	}
}
__global__ void CompareKernel(float* Data, int* Result, float Th, char CompareFlag, unsigned int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		if (CompareFlag == '<')
		{
			if (Data[idx] < Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '>')
		{
			if (Data[idx] > Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '<=')
		{
			if (Data[idx] <= Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '>=')
		{
			if (Data[idx] >= Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '==')
		{
			if (Data[idx] == Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
	}
}
__global__ void CompareKernel(hipComplex* Data, int* Result, float Th, char CompareFlag, unsigned int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		if (CompareFlag == '<')
		{
			if (hipCabsf(Data[idx]) < Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '>')
		{
			if (hipCabsf(Data[idx]) > Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '<=')
		{
			if (hipCabsf(Data[idx]) <= Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '>=')
		{
			if (hipCabsf(Data[idx]) >= Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
		else if (CompareFlag == '==')
		{
			if (hipCabsf(Data[idx]) == Th)
			{
				Result[idx] = 1;
			}
			else
			{
				Result[idx] = 0;
			}
		}
	}
}

/*
���
Data: [R C B]
Result: [1 C B]
*/
__global__ void cuSumKernel(float *Data, float* Result, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows * Cols * Bands)
	{
		/*resultƫ��*/
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		atomicAdd(&Result[B_bias * Cols + C_bias], Data[idx]);
	}
}
__global__ void cuSumKernel(hipComplex* Data, hipComplex* Result, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows * Cols * Bands)
	{
		/*resultƫ��*/
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		atomicAdd(&Result[B_bias * Cols + C_bias].x, Data[idx].x);
		atomicAdd(&Result[B_bias * Cols + C_bias].y, Data[idx].y);
	}

}
__global__ void cuSumKernel(int *Data, int* Result, unsigned int Rows, unsigned int Cols, unsigned int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows * Cols * Bands)
	{
		/*resultƫ��*/
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		atomicAdd(&Result[B_bias * Cols + C_bias], Data[idx]);
	}
}

__global__ void cuComplexPowATileMean(hipComplex *data, hipComplex *atilemean, hipComplex *sum_mean, float *powatilemean, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		float mean1 = sum_mean[0].x / N;
		float mean2 = sum_mean[0].y / N;
		float dataI = data[idx].x - mean1;//�ظ�
		float dataQ = data[idx].y - mean2;//�ظ�
		atilemean[idx].x = dataI;//�ظ�
		atilemean[idx].y = dataQ;//�ظ�
		powatilemean[idx] = dataI * dataI + dataQ * dataQ;
	}
}

__global__ void SqrtTileComplexDiv(float *sum, hipComplex *alitemean, hipComplex *result, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		float temp = sum[0] / (N - 1);
		float sigmma = sqrt(temp);
		result[idx].x = alitemean[idx].x / sigmma;
		result[idx].y = alitemean[idx].y / sigmma;
	}
}
__global__ void SqrtTileDiv(float *sum, float *alitemean, float *result, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		float temp = sum[0] / (N - 1);
		float sigmma = sqrt(temp);
		result[idx] = alitemean[idx] / sigmma;
	}
}
__global__ void cuPowATileMean(float *data, float *atilemean, float *sum_mean, float *powatilemean, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		float mean = sum_mean[0] / N;
		float tmp = data[idx] - mean;//�ظ�
		atilemean[idx] = tmp;
		powatilemean[idx] = tmp * tmp;
	}
}

__global__ void z_score_kernel(hipComplex *data, hipComplex *atiledata, float* PowerResult, unsigned int Rows, unsigned int Cols, unsigned int Bands, hipComplex *Sum_first, float *Sum_second, hipComplex* z_score)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Rows * Cols * Bands)
	{
		unsigned int N = Rows*Cols*Bands;
		/*Sum_firstƫ��*/
		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;

		//ԭ�Ӳ������
		atomicAdd(&Sum_first[B_bias * Cols + C_bias].x, data[idx].x);
		atomicAdd(&Sum_first[B_bias * Cols + C_bias].y, data[idx].y);

		
		float mean1 = Sum_first[0].x / N;
		float mean2 = Sum_first[0].y / N;
		atiledata[idx].x = data[idx].x - mean1;//�ظ�
		atiledata[idx].y = data[idx].y - mean2;
		PowerResult[idx] = (atiledata[idx].x) *(atiledata[idx].x) + (atiledata[idx].y)*(atiledata[idx].y);
		atomicAdd(&Sum_second[B_bias * Cols + C_bias], PowerResult[idx]);

		float temp = Sum_second[0] / (N - 1);
		float sigmma = sqrt(temp);
		z_score[idx].x = (atiledata[idx].x) / sigmma;
		z_score[idx].y = (atiledata[idx].y) / sigmma;
	}
}

// ���н���3άѭ����λ
__global__ void CirculShiftCol(float* Data, float* Result, int shift_num, int Rows, int Cols, int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	unsigned int RCMul = Rows * Cols;
	if (idx < RCMul * Bands)
	{
		// ��ά����
		unsigned int indx = (idx % RCMul) % Rows;
		unsigned int indy = (idx % RCMul) / Rows;
		unsigned int indz = idx / RCMul;

		int indx_shift = (indx + Rows + shift_num) % Rows;

		int index_new = indz * RCMul + indy * Rows + indx_shift;

		Result[idx] = Data[index_new];
	}
}
__global__ void CirculShiftCol(hipComplex* Data, hipComplex* Result, int shift_num, int Rows, int Cols, int Bands)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	unsigned int RCMul = Rows * Cols;
	if (idx < RCMul * Bands)
	{
		// ��ά����
		unsigned int indx = (idx % RCMul) % Rows;
		unsigned int indy = (idx % RCMul) / Rows;
		unsigned int indz = idx / RCMul;

		int indx_shift = (indx + Rows + shift_num) % Rows;

		int index_new = indz * RCMul + indy * Rows + indx_shift;

		Result[idx] = Data[index_new];
	}
}

// ����2λѭ����λ
__global__ void Circcushift(hipComplex *src, hipComplex *tar, int  Row, int Col, int shift_num)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int shift = (idx + shift_num * Row) % (Row * Col);

	if (idx < Row * Col)
	{
		tar[idx] = src[shift];
	}
}
__global__ void Circcushift(float *src, float *tar, int  Row, int Col, int shift_num)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int shift = (idx + shift_num * Row) % (Row * Col);

	if (idx < Row * Col)
	{
		tar[idx] = src[shift];
	}
}




/*�ź�����ר��*/
__global__ void SetPerTarHn(hipComplex *Data, hipComplex *Temp, int colIndex, int elements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < elements)
	{
		Data[colIndex * elements + i] = Temp[i];
	}
}