#include "hip/hip_runtime.h"
#include "tools.cuh"
#include "CommonKernel.cuh"
#include "hipblas.h"

namespace tools
{
	/*���������еĸ�������*/
	void SaveHostComplexMatrix(hipComplex *h_data, int rows, int cols, string path)
	{
		ofstream fp;
		fp.open(path, ios::out | ios::trunc);
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
			{
				float RData = h_data[int(j * rows + i)].x;
				float IData = h_data[int(j * rows + i)].y;
				// д��ʵ��
				fp << RData;
				// д����ź��鲿
				if (IData != -0 || IData != 0)
				{
					fp << (IData > 0 ? "+" : "") << IData << "i";
				}
				// д�����ϵļ��
				if (j != cols - 1)
				{
					fp << " ";
				}
			}
			fp << endl;
		}
		fp << endl;
		fp.close();
		std::cout << path << "	Matrix Save sucess!" << endl;

	}

	/*�����豸�еĸ�������*/
	void SaveDeviceComplexMatrix(hipComplex *d_data, int rows, int cols, string path)
	{
		hipDeviceSynchronize();
		hipComplex *h_data = (hipComplex *)malloc(sizeof(hipComplex) * rows * cols);
		hipMemcpy(h_data, d_data, sizeof(hipComplex) * rows * cols, hipMemcpyDeviceToHost);

		ofstream fp;
		fp.open(path, ios::out | ios::trunc);
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
			{
				float RData = h_data[int(j * rows + i)].x;
				float IData = h_data[int(j * rows + i)].y;
				// д��ʵ��
				fp << RData;
				// д����ź��鲿
				if (IData != -0 || IData != 0)
				{
					fp << (IData > 0 ? "+" : "") << IData << "i";
				}
				// д�����ϵļ��
				if (j != cols - 1)
				{
					fp << " ";
				}
			}
			fp << endl;
		}
		fp << endl;
		fp.close();
		std::cout << path << "	Matrix Save sucess!" << endl;
	}

	/*�����豸�е�ʵ������*/
	void SaveDeviceFloatMatrix(float* d_data, int rows, int cols, string path)
	{
		hipDeviceSynchronize();
		float* h_data = (float*)malloc(sizeof(float) * rows * cols);
		hipMemcpy(h_data, d_data, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);

		ofstream fp;
		fp.open(path, ios::out | ios::trunc);
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
			{
				float Data = h_data[int(j * rows + i)];

				fp << Data;

				// д�����ϵļ��
				if (j != cols - 1)
				{
					fp << " ";
				}
			}
			fp << endl;
		}
		fp << endl;
		fp.close();
		std::cout << path << "	Matrix Save sucess!" << endl;
	}

	/*�����豸�е���������*/
	void SaveDeviceIntMatrix(int *d_data, int rows, int cols, string path)
	{
		hipDeviceSynchronize();
		int *h_data = (int *)malloc(sizeof(int) * rows * cols);
		hipMemcpy(h_data, d_data, sizeof(int) * rows * cols, hipMemcpyDeviceToHost);

		ofstream fp;
		fp.open(path, ios::out | ios::trunc);
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
			{
				// д��ʵ��
				fp << h_data[int(j * rows + i)];

				// д�����ϵļ��
				if (j != cols - 1)
				{
					fp << " ";
				}
			}
			fp << endl;
		}
		fp << endl;
		fp.close();
		std::cout << path << "	Matrix Save sucess!" << endl;
	}


	/*���������еĸ�������*/
	void SaveHostComplexVector(hipComplex *h_data, int elements, std::string path)
	{
		
		std::fstream fp;
		fp.open(path, std::ios::in | std::ios::out | std::ios::trunc);
		for (int i = 0; i < elements; i++)
		{
			float RData = h_data[i].x;
			float IData = h_data[i].y;
			// д��ʵ��
			fp << RData;
			// д����ź��鲿
			if (IData != -0 || IData != 0)
			{
				fp << (IData > 0 ? "+" : "") << IData << "i";
			}
			fp << endl;
		}
		fp.close();
		std::cout << path << " Save sucess!" << endl;
	}

	/*�����豸�еĸ�������*/
	void SaveDeviceComplexVector(hipComplex *d_data, int elements, std::string path)
	{
		hipDeviceSynchronize();
		hipComplex *h_data = (hipComplex *)malloc(sizeof(hipComplex) * elements);
		hipMemcpy(h_data, d_data, sizeof(hipComplex) * elements, hipMemcpyDeviceToHost);

		std::fstream fp;
		fp.open(path, std::ios::in | std::ios::out | std::ios::trunc);
		for (int i = 0; i < elements; i++)
		{
			float RData = h_data[i].x;
			float IData = h_data[i].y;
			// д��ʵ��
			fp << RData;
			// д����ź��鲿
			if (IData != -0 || IData != 0)
			{
				fp << (IData > 0 ? "+" : "") << IData << "i";
			}
			fp << endl;
		}
		fp.close();
		std::cout << path << " Save sucess!" << endl;
	}
	/*�����豸�еĸ�������*/
	void SaveDeviceIntVector(int* d_data, int elements, std::string path)
	{
		hipDeviceSynchronize();
		int* h_data = (int*)malloc(sizeof(int) * elements);
		hipMemcpy(h_data, d_data, sizeof(int) * elements, hipMemcpyDeviceToHost);

		std::fstream fp;
		fp.open(path, std::ios::in | std::ios::out | std::ios::trunc);
		for (int i = 0; i < elements; i++)
		{
			int Data = h_data[i];
			fp << Data;
			fp << endl;
		}
		fp.close();
		std::cout << path << " Save sucess!" << endl;
	}

	/*�����豸�е�����*/
	void SaveDeviceFloatVector(float *d_data, int elements, std::string path)
	{
		hipDeviceSynchronize();
		float *h_data = (float *)malloc(sizeof(float) * elements);
		hipMemcpy(h_data, d_data, sizeof(float) * elements, hipMemcpyDeviceToHost);

		std::fstream fp;
		fp.open(path, std::ios::in | std::ios::out | std::ios::trunc);
		for (int i = 0; i < elements; i++)
		{
			float RData = h_data[i];
			// д��ʵ��
			fp << RData;
			fp << endl;
		}
		fp.close();
		std::cout << path << " Save sucess!" << endl;
	}


	/*���������е�short������*/
	void SaveHostShortVector(short* h_data, int elements, std::string path)
	{
		hipDeviceSynchronize();

		std::fstream fp;
		fp.open(path, std::ios::in | std::ios::out | std::ios::trunc);
		for (int i = 0; i < elements; i++)
		{
			float RData = h_data[i];
			// д��ʵ��
			fp << RData;
			fp << endl;
		}
		fp.close();
		std::cout << path << " Save sucess!" << endl;
	}

	/*���������еĸ�������*/
	template <typename Type>
	void SaveHostVector(Type *d_data, int elements, string path)
	{
		std::fstream fp;
		fp.open(path, std::ios::in | std::ios::out | std::ios::trunc);
		for (int i = 0; i < elements; i++)
		{
			// д��ʵ��
			fp << d_data[i];
			fp << endl;
		}
		fp.close();
		std::cout << path << " Save sucess!" << endl;
	}

	void Zeros(CudaArray<hipComplex> &data, unsigned int N, hipStream_t stream)
	{
		data.Resize(N, 1, 1);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		hipComplex Temp;Temp.x = 0.0;Temp.y = 0.0;
		ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data.FirstAddr(), Temp, N);
	}
	void Zeros(CudaArray<float> &data, unsigned int N, hipStream_t stream)
	{
		data.Resize(N, 1, 1);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data.FirstAddr(), 0.0, N);
	}

	/*dB Result = 20*log10(Data)*/
	FCMat dB(FCMat& Data, hipStream_t stream)
	{
		unsigned int Size = Data.elements();
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;

		FCMat Result(Data.dims(0), Data.dims(1), Data.dims(2));
		dBKernel << <blocksPerGrid, threadsPerBlock >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

		return Result;
	}

	/*idB Result = pow(10, Data / 20)*/
	FCMat idB(FCMat& Data, hipStream_t stream)
	{
		unsigned int Size = Data.elements();
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;

		FCMat Result(Data.dims(0), Data.dims(1), Data.dims(2));
		idBKernel << <blocksPerGrid, threadsPerBlock >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

		return Result;
	}

	/*��ʵ������תΪ�鲿Ϊ0�ĸ�������*/
	CCMat Complex(FCMat& Data, hipStream_t stream)
	{
		unsigned int Size = Data.elements();
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;

		CCMat Result(Data.dims(0), Data.dims(1), Data.dims(2));
		ComplexKernel << <blocksPerGrid, threadsPerBlock >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

		return Result;
	}
	/*��ʵ������תΪ�鲿Ϊ0�ĸ�������*/
	void Complex(FCMat& Data, CCMat& Result, hipStream_t stream)
	{
		unsigned int Size = Data.elements();
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;

		Result.Resize(Data.dims(0), Data.dims(1), Data.dims(2));
		ComplexKernel << <blocksPerGrid, threadsPerBlock >> > (Data.FirstAddr(), Result.FirstAddr(), Size);


	}


	/*������ʵ������תΪ��������*/
	CCMat Complex(FCMat& Real, FCMat& Imag, hipStream_t stream)
	{
		unsigned int RealSize = Real.elements();
		unsigned int ImagSize = Imag.elements();
		if (RealSize != ImagSize)
		{
			printf("ά�Ȳ�һ��\n");throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (RealSize + threadsPerBlock - 1) / threadsPerBlock;

		CCMat Result(Real.dims(0), Real.dims(1), Real.dims(2));
		ComplexMat << <blocksPerGrid, threadsPerBlock >> > (Result.FirstAddr(), Real.FirstAddr(), Imag.FirstAddr(), RealSize);

		return Result;
	}

	/*������ʵ������תΪ��������*/
	void Complex(FCMat& Real, FCMat& Imag, CCMat& Result,  hipStream_t stream)
	{
		unsigned int RealSize = Real.elements();
		unsigned int ImagSize = Imag.elements();
		if (RealSize != ImagSize)
		{
			printf("ά�Ȳ�һ��\n"); throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (RealSize + threadsPerBlock - 1) / threadsPerBlock;

		Result.Resize(Real.dims(0), Real.dims(1), Real.dims(2));
		ComplexMat << <blocksPerGrid, threadsPerBlock >> > (Result.FirstAddr(), Real.FirstAddr(), Imag.FirstAddr(), RealSize);


	}

	FCMat Real(CCMat& Data, hipStream_t Stream)
	{
		unsigned int Size = Data.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
		FCMat Result(Data.dims(0), Data.dims(1), Data.dims(2));
		Realkernel << <blocksPerGrid, threadsPerBlock, 0, Stream >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

		return Result;
	}

	void Real(CCMat& Data, FCMat& Result, hipStream_t Stream)
	{
		unsigned int Size = Data.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
		Result.Resize(Data.dims(0), Data.dims(1), Data.dims(2));
		Realkernel << <blocksPerGrid, threadsPerBlock, 0, Stream >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

	}

	FCMat Imag(CCMat& Data, hipStream_t Stream)
	{
		unsigned int Size = Data.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
		FCMat Result(Data.dims(0), Data.dims(1), Data.dims(2));
		Imagkernel << <blocksPerGrid, threadsPerBlock, 0, Stream >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

		return Result;
	}

	void Imag(CCMat& Data, FCMat& Result, hipStream_t Stream)
	{
		unsigned int Size = Data.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
		Result.Resize(Data.dims(0), Data.dims(1), Data.dims(2));
		Imagkernel << <blocksPerGrid, threadsPerBlock, 0, Stream >> > (Data.FirstAddr(), Result.FirstAddr(), Size);

	}


	/*����������*/
	CCMat DotMul(CCMat& Data1, CCMat& Data2, hipStream_t stream)
	{
		CCMat Result(Data1.dims(0), Data1.dims(1), Data1.dims(2));
		unsigned int Data1Size = Data1.elements();
		unsigned int Data2Size = Data2.elements();
		if (Data1Size != Data2Size)
		{
			printf("ά�Ȳ�һ��, Data1: %d, Data2: %d\n", Data1Size, Data2Size);
			throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		DotMulKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2.FirstAddr(), Result.FirstAddr(), Data1Size);

		return Result;
	}

	/*����������*/
	void StillDotMul(CCMat& Data1, CCMat& Data2, hipStream_t stream)
	{
		unsigned int Data1Size = Data1.elements();
		unsigned int Data2Size = Data2.elements();
		if (Data1Size != Data2Size)
		{
			printf("ά�Ȳ�һ��, Data1: %d, Data2: %d\n", Data1Size, Data2Size);
			throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		DotMul2Kernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2.FirstAddr(), Data1Size);

	}

	/*��������ӷ�*/
	CCMat MatAdd(CCMat& Data1, CCMat& Data2, hipStream_t stream)
	{
		CCMat Result(Data1.dims(0), Data1.dims(1), Data1.dims(2));
		unsigned int Data1Size = Data1.elements();
		unsigned int Data2Size = Data2.elements();
		if (Data1Size != Data2Size)
		{
			printf("��������ӷ� ʱ ά�Ȳ�һ��\n"); throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		MatAddKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2.FirstAddr(), Result.FirstAddr(), Data1Size);

		return Result;
	}

	/*��������ӷ�*/
	void MatAdd(CCMat& Data1, CCMat& Data2, CCMat& Result, hipStream_t stream)
	{
		Result.Resize(Data1.dims(0), Data1.dims(1), Data1.dims(2));

		unsigned int Data1Size = Data1.elements();
		unsigned int Data2Size = Data2.elements();
		if (Data1Size != Data2Size)
		{
			printf("��������ӷ� ʱ ά�Ȳ�һ��\n"); throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		MatAddKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2.FirstAddr(), Result.FirstAddr(), Data1Size);

	}

	/*��������ӷ�*/
	void MatAdd(CCMat& Data1, CCMat& Data2, CCMat& Data3, CCMat& Result, hipStream_t stream)
	{
		Result.Resize(Data1.dims(0), Data1.dims(1), Data1.dims(2));

		unsigned int Data1Size = Data1.elements();
		unsigned int Data2Size = Data2.elements();
		if (Data1Size != Data2Size)
		{
			printf("��������ӷ� ʱ ά�Ȳ�һ��\n"); throw;
		}

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		MatAddKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2.FirstAddr(), Result.FirstAddr(), Data1Size);

	}

	/*��������ӷ�*/
	void MatAdd(CCMat& Data1, hipComplex Data2, hipStream_t stream)
	{

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1.elements() + threadsPerBlock - 1) / threadsPerBlock;

		MatAddComplexKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Data1.elements());

	}

	/*float����˷�*/
	FCMat MatMul(FCMat& A, FCMat& B, hipStream_t stream)
	{
		if (A.dims(1) != B.dims(0))
		{
			printf("ά�Ȳ�һ��");throw;
		}

		FCMat C(A.dims(0), B.dims(1), 1);

		int A_Row = A.dims(0);
		int A_Col = A.dims(1);
		int B_Row = B.dims(0);
		int B_Col = B.dims(1);

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		float a = 1, b = 0;
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A_Row, B_Col, A_Col, &a, A.FirstAddr(), A_Row, B.FirstAddr(), B_Row, &b, C.FirstAddr(), A_Row);

		hipblasDestroy(handle);

		return C;
	}

	
	/*��������˷�*/
	CCMat MatMul(CCMat& A, CCMat& B, hipStream_t stream)
	{
		if (A.dims(1) != B.dims(0))
		{
			printf("ά�Ȳ�һ��");throw;
		}

		CCMat C(A.dims(0), B.dims(1), 1);

		int A_Row = A.dims(0);
		int A_Col = A.dims(1);
		int B_Row = B.dims(0);
		int B_Col = B.dims(1);

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipComplex a,b;
		a.x = 1;a.y = 0;
		b.x = 0;b.y = 0;
		hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A_Row, B_Col, A_Col, &a, A.FirstAddr(), A_Row, B.FirstAddr(), B_Row, &b, C.FirstAddr(), A_Row);

		hipblasDestroy(handle);
		return C;
	}

	/*��������˷�*/
	void MatMul(CCMat& A, CCMat& B, CCMat& C, hipStream_t stream)
	{
		if (A.dims(1) != B.dims(0))
		{
			printf("ά�Ȳ�һ��"); throw;
		}

		C.Resize(A.dims(0), B.dims(1), 1);

		int A_Row = A.dims(0);
		int A_Col = A.dims(1);
		int B_Row = B.dims(0);
		int B_Col = B.dims(1);

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipComplex a, b;
		a.x = 1; a.y = 0;
		b.x = 0; b.y = 0;
		hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A_Row, B_Col, A_Col, &a, A.FirstAddr(), A_Row, B.FirstAddr(), B_Row, &b, C.FirstAddr(), A_Row);

		hipblasDestroy(handle);

	}

	/*���������float*/
	CCMat CFDiv(CCMat& Data1, float Data2, hipStream_t stream)
	{
		CCMat Result(Data1.dims(0), Data1.dims(1), Data1.dims(2));
		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		C2FDivKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Result.FirstAddr(), Data1Size);

		return Result;
	}

	/*ԭ�ظ��������float*/
	void StillCFDiv(CCMat& Data1, float Data2, hipStream_t stream)
	{
		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		C2FDiv2Kernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Data1Size);

	}



	/*���������float*/
	void MatFMul(CCMat& Data1, FCMat& Data2, hipStream_t stream)
	{
		if (Data1.elements() != Data2.elements()) return;

		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		MatMulKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2.FirstAddr(), Data1Size);

	}

	/*���������float*/
	CCMat MatFMul(CCMat& Data1, float Data2, hipStream_t stream)
	{
		CCMat Result(Data1.dims(0), Data1.dims(1), Data1.dims(2));
		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		C2FMulKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Result.FirstAddr(), Data1Size);

		return Result;
	}

	/*���������float*/
	void MatFMul(CCMat& Data1, CCMat& Result, float Data2, hipStream_t stream)
	{
		Result.Resize(Data1.dims(0), Data1.dims(1), Data1.dims(2));
		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		C2FMulKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Result.FirstAddr(), Data1Size);

	}

	/*���������float*/
	void StillMatFMul(CCMat& Data1, float Data2, hipStream_t stream)
	{
		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		C2FMulKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Data1.FirstAddr(), Data1Size);

	}


	/*float�����float*/
	FCMat MatFMul(FCMat& Data1, float Data2, hipStream_t stream)
	{
		FCMat Result(Data1.dims(0), Data1.dims(1), Data1.dims(2));
		unsigned int Data1Size = Data1.elements();

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data1Size + threadsPerBlock - 1) / threadsPerBlock;

		F2FMulKernal << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data1.FirstAddr(), Data2, Result.FirstAddr(), Data1Size);

		return Result;
	}


	/*��ȡ���*/
	CCMat Slice(CCMat& Data, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		CCMat Result(REnd - RStart + 1, CEnd - CStart + 1, BEnd - BStart + 1);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows*Cols*Bands + threadsPerBlock - 1) / threadsPerBlock;

		SliceKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Result.FirstAddr(), RStart, REnd, CStart, CEnd, BStart, BEnd, Rows, Cols, Bands);

		return Result;
	}

	/*��ȡ���(FCMat)*/
	FCMat Slice(FCMat& Data, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		FCMat Result(REnd - RStart + 1, CEnd - CStart + 1, BEnd - BStart + 1);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows*Cols*Bands + threadsPerBlock - 1) / threadsPerBlock;

		SliceKernel << <blocksPerGrid, threadsPerBlock >> > (Data.FirstAddr(), Result.FirstAddr(), RStart, REnd, CStart, CEnd, BStart, BEnd, Rows, Cols, Bands);

		return Result;
	}

	/*��ȡ���(FCMat)*/
	void Slice(FCMat& Data, FCMat& Result, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		Result.Resize(REnd - RStart + 1, CEnd - CStart + 1, BEnd - BStart + 1);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows*Cols*Bands + threadsPerBlock - 1) / threadsPerBlock;

		SliceKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Result.FirstAddr(), RStart, REnd, CStart, CEnd, BStart, BEnd, Rows, Cols, Bands);

	}

	/*��ȡ���*/
	void Slice(CCMat& Data, CCMat& Result, int RStart, int REnd, int CStart, int CEnd, int BStart, int BEnd, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		Result.Resize(REnd - RStart + 1, CEnd - CStart + 1, BEnd - BStart + 1);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows*Cols*Bands + threadsPerBlock - 1) / threadsPerBlock;

		SliceKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Result.FirstAddr(), RStart, REnd, CStart, CEnd, BStart, BEnd, Rows, Cols, Bands);

	}


	/*��������չ
		��������Data��չΪResult���ȣ�������չ��
		Data : (Rows, Cols, Bands)
		Result : (Rows + abs(AddLength), Cols, Bands)
		AddLength : ��չ�������Ҹ���
		Value : ��չ��ֵ
	*/
	FCMat cuExtend(FCMat& Data, int AddLength, float Value, hipStream_t stream)
	{
		if (AddLength == 0) return Data;

		FCMat Result(Data.dims(0) + abs(AddLength), Data.dims(1), Data.dims(2));
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Result.elements() + threadsPerBlock - 1) / threadsPerBlock;

		ExtendKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Result.FirstAddr(), AddLength, Value, Data.dims(0), Data.dims(1), Data.dims(2));

		return Result;
	}

	/*��������չ
	��������Data��չΪResult���ȣ�������չ��
	Data : (Rows, Cols, Bands)
	Result : (Rows + abs(AddLength), Cols, Bands)
	AddLength : ��չ�������Ҹ���
	Value : ��չ��ֵ
	*/
	CCMat cuExtend(CCMat& Data, int AddLength, hipComplex Value, hipStream_t stream)
	{
		if (AddLength == 0) return Data;

		CCMat Result(Data.dims(0) + abs(AddLength), Data.dims(1), Data.dims(2));
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Result.elements() + threadsPerBlock - 1) / threadsPerBlock;

		ExtendKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Result.FirstAddr(), AddLength, Value, Data.dims(0), Data.dims(1), Data.dims(2));

		return Result;
	}

	void cuExtend(CCMat& Data, CCMat& Result, int AddLength, hipComplex Value, hipStream_t stream)
	{
		if (AddLength == 0)
		{
			Result.CopyFromDevice(Data);
			return;
		}

		Result.Resize(Data.dims(0) + abs(AddLength), Data.dims(1), Data.dims(2));
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Result.elements() + threadsPerBlock - 1) / threadsPerBlock;

		ExtendKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Result.FirstAddr(), AddLength, Value, Data.dims(0), Data.dims(1), Data.dims(2));

	}

	// ת��
	void Transpose(FCMat& data, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		FCMat result(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		TransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		hipDeviceSynchronize();
		data = result;

		result.Free();
	};

	// ת��
	FCMat cuTranspose(FCMat& data, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		FCMat result(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		TransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		
		return result;
	};

	// ת��
	void Transpose(CCMat& data, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		CCMat result(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		TransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		
		data.CopyFromDevice(result);

		result.Free();
	};

	// ת��
	CCMat cuTranspose(CCMat& data, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		CCMat result(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		TransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		
		return result;
	};
void Transpose(CCMat& data, CCMat& result, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		result.Resize(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		TransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);

		/*data.CopyFromDevice(result);

		result.Free();*/
	};

	// ����ת��
	void CTranspose(CCMat& data, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		CCMat result(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		CTransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		hipDeviceSynchronize();
		data = result;

		result.Free();
	};

	// ����ת��
	CCMat cuCTranspose(CCMat& data, hipStream_t stream)
	{
		int Rows = data.dims(0);
		int Cols = data.dims(1);
		int Bands = data.dims(2);

		CCMat result(Cols, Rows, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;

		CTransposeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);

		return result;
	};

	// ���
	float cuSum(FCMat& Data, hipStream_t stream)
	{
		float result;
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipblasSasum(handle, Data.elements(), Data.FirstAddr(), 1, &result);
		hipblasDestroy(handle);
		return result;
	}

	// ����abs���
	float cuSum(CCMat& Data, hipStream_t stream)
	{
		float result;
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipblasScasum(handle, Data.elements(), Data.FirstAddr(), 1, &result);
		hipblasDestroy(handle);
		return result;
	}

	// ��ȡ���ֵ
	float cuMax(FCMat& Data, hipStream_t stream)
	{
		int idx;
		int Rows =  Data.dims(0);
		int Cols =  Data.dims(1);
		int Bands = Data.dims(2);

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipblasIsamax(handle, Data.elements(), Data.FirstAddr(), 1, &idx);
		hipblasDestroy(handle);

		// ������
		unsigned int B_bias = idx / (Rows * Cols);
		// ������
		unsigned int C_bias = (idx % (Rows * Cols)) / Rows;
		// ������
		unsigned int R_bias = (idx % (Rows * Cols)) % Rows;
		
		float result;
		// ��ȡ����Ԫ�ص�ֵ
		hipMemcpy(&result, Data.FirstAddr() + idx, sizeof(float), hipMemcpyDeviceToHost);

		return result;
	}

	// ��ȡ���ֵ���±�
	void cuMax(FCMat& Data, float& result, dim3& index, hipStream_t stream)
	{
		int idx;
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipblasIsamax(handle, Data.elements(), Data.FirstAddr(), 1, &idx);
		hipblasDestroy(handle);

		idx = idx - 1;

		// ������
		index.z = idx / (Rows * Cols);
		// ������
		index.y = (idx % (Rows * Cols)) / Rows;
		// ������
		index.x = (idx % (Rows * Cols)) % Rows;

		// ��ȡ����Ԫ�ص�ֵ
		hipMemcpy(&result, Data.FirstAddr() + idx, sizeof(float), hipMemcpyDeviceToHost);
	}

	// ��ȡ���ֵ���±�
	void cuMax(FCMat& Data, FCMat& result, dim3& index, hipStream_t stream)
	{
		int idx;
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetStream(handle, stream);
		hipblasIsamax(handle, Data.elements(), Data.FirstAddr(), 1, &idx);
		hipblasDestroy(handle);

		idx = idx - 1;

		// ������
		index.z = idx / (Rows * Cols);
		// ������
		index.y = (idx % (Rows * Cols)) / Rows;
		// ������
		index.x = (idx % (Rows * Cols)) % Rows;

		// ��ȡ����Ԫ�ص�ֵ
		result.CopyFromDevicePoint(Data.FirstAddr() + idx, 1, 1, 1);
	}

	/*float���������*/
	FCMat sum(FCMat& Data, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		FCMat result(1, Cols, Bands, 0.0);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		cuSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		return result;
	}

	/*float���������*/
	void sum(FCMat& Data, FCMat& result, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		result.Resize(1, Cols, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(result.FirstAddr(), 0.0, Cols * Bands);

		blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		cuSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);

	}

	/*�������������*/
	CCMat sum(CCMat& Data, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		hipComplex temp; temp.x = 0.0; temp.y = 0.0;
		CCMat result(1, Cols, Bands, temp);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		cuSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		return result;
	}
	/*�������������*/
	void sum(CCMat& Data, CCMat& result, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		hipComplex temp; temp.x = 0.0; temp.y = 0.0;
		result.Resize(1, Cols, Bands, temp, stream);
		
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(result.FirstAddr(), temp, Cols * Bands);

		blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		cuSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
	}
	/*ʵ�����������*/
	ICMat sum(ICMat& Data, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		ICMat result(1, Cols, Bands);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		cuSumKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), result.FirstAddr(), Rows, Cols, Bands);
		return result;
	}

	// ��һ��
	void Z_ScoreStandardization(FCMat &data, FCMat &z_score, int N, hipStream_t stream)
	{
		FCMat SumValue(1, 1, 1, 0.0, stream);
		sum(data, SumValue, stream);
		FCMat atiledata(N, 1, 1);

		int threadsPerBlock = 1024;
		unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		FCMat powatilemean(N, 1, 1);
		cuPowATileMean << <blocksPerGrid, threadsPerBlock, 0, stream>> >(data.FirstAddr(), atiledata.FirstAddr(), SumValue.FirstAddr(), powatilemean.FirstAddr(), N);

		FCMat Sum(1, 1, 1, 0.0, stream);
		sum(powatilemean, Sum, stream);

		z_score.Resize(N, 1, 1);
		//����Ϊ�˻�ȡSum���Ա������sigma
		SqrtTileDiv << <blocksPerGrid, threadsPerBlock, 0, stream>> >(Sum.FirstAddr(), atiledata.FirstAddr(), z_score.FirstAddr(), N);

		//�ͷſռ�
		SumValue.Free();
		atiledata.Free();
		powatilemean.Free();
		Sum.Free();
	}

	void Z_ScoreStandardization(CCMat &data, CCMat &z_score, int N, hipStream_t stream)
	{
		//�������
		hipComplex initValue;	initValue.x = 0.0;	initValue.y = 0.0;
		CCMat SumValue(1,1,1, initValue, stream);
		sum(data, SumValue, stream);

		CCMat atiledata(N, 1, 1);
		int threadsPerBlock = 1024;
		unsigned int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		FCMat PowerResult(N, 1, 1);

		cuComplexPowATileMean << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data.FirstAddr(), atiledata.FirstAddr(), SumValue.FirstAddr(), PowerResult.FirstAddr(), N);

		FCMat Sum(1,1,1,0.0,stream);
		sum(PowerResult, Sum, stream);

		z_score.Resize(N, 1, 1);
		//����Ϊ�˻�ȡSum���Ա������sigma
		SqrtTileComplexDiv << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Sum.FirstAddr(), atiledata.FirstAddr(),
					z_score.FirstAddr(), N);

		//�ͷſռ�
		SumValue.Free();
		atiledata.Free();
		PowerResult.Free();
		Sum.Free();
	
		/******************************************************/
	}

	void Z_ScoreStandardization_con(CCMat &data, CCMat &z_score, int N, hipStream_t stream)
	{
		//hipComplex initialValue;	initialValue.x = 0.0; initialValue.y = 0.0;
		z_score.Resize(data.dims(0), data.dims(1), data.dims(2));
		unsigned int Rows = data.dims(0);
		unsigned int Cols = data.dims(1);
		unsigned int Bands = data.dims(2);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
		hipComplex temp; temp.x = 0.0, temp.y = 0.0;
		CCMat Sum_first(1, 1, 1, temp, stream);
		CCMat atiledata(N, 1, 1);
		FCMat PowerResult(N, 1, 1);
		FCMat Sum_second(1, 1, 1, temp.x, stream);
		z_score_kernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (data.FirstAddr(), atiledata.FirstAddr(), PowerResult.FirstAddr(), Rows, Cols, Bands, Sum_first.FirstAddr(), Sum_second.FirstAddr(), z_score.FirstAddr());

		Sum_first.Free();
		atiledata.Free();
		PowerResult.Free();
		Sum_second.Free();
	}

	// FFT
	/*��չ��FFTNum�����FFT ����*/
	CCMat cuFFT(CCMat &Data, unsigned int FFTNum, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		CCMat DataF;
		hipComplex Value; Value.x = 0.0; Value.y = 0.0;
		// ��Ҫ��չ����չ
		if (FFTNum > Rows)
		{
			// ��fft������չ�����У�
			cuExtend(Data, DataF, FFTNum - Rows, Value, stream);

		}
		else
		{
			DataF.CopyFromDevice(Data);
			FFTNum = Rows;
		}

		hipfftHandle Handle;
		hipfftPlan1d(&Handle, FFTNum, HIPFFT_C2C, Cols);
		hipfftSetStream(Handle, stream);
		for (int i = 0; i < Bands; i++)
		{
			hipfftExecC2C(Handle, DataF.FirstAddr() + i * (FFTNum * Cols), DataF.FirstAddr() + i * (FFTNum * Cols), HIPFFT_FORWARD);
		}
		hipfftDestroy(Handle);

		return DataF;
	}
	void cuFFT(CCMat &Data, CCMat& DataF, unsigned int FFTNum, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		
		hipComplex Value; Value.x = 0.0; Value.y = 0.0;
		// ��Ҫ��չ����չ
		//if (FFTNum > Rows)
		//{
		//	// ��fft������չ�����У�
		//	cuExtend(Data, DataF, FFTNum - Rows, Value, stream);

		//}
		//else
		//{
		//	DataF.CopyFromDevice(Data);
		//	FFTNum = Rows;
		//}

		cuExtend(Data, DataF, FFTNum - Rows, Value, stream);

		//CCMat DataTemp(DataF.dims(0), DataF.dims(1), DataF.dims(2));

		hipfftHandle Handle;
		hipfftPlan1d(&Handle, FFTNum, HIPFFT_C2C, Cols);
		hipfftSetStream(Handle, stream);
		for (int i = 0; i < Bands; i++)
		{
			//hipfftExecC2C(Handle, DataF.FirstAddr() + i * (FFTNum * Cols), DataTemp.FirstAddr() + i * (FFTNum * Cols), HIPFFT_FORWARD);
			hipfftExecC2C(Handle, DataF.FirstAddr() + i * (FFTNum * Cols), DataF.FirstAddr() + i * (FFTNum * Cols), HIPFFT_FORWARD);
		}
		hipfftDestroy(Handle);

		//DataF.CopyFromDevice(DataTemp);

		//DataTemp.Free();


	}

	// IFFT
	/*IFFT���ٽضϵ�OriginNum ����*/
	void cuIFFT(CCMat &Data, unsigned int OriginNum, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);

		CCMat DataTemp(Data.dims(0), Data.dims(1), Data.dims(2));

		hipfftHandle plan;
		hipfftPlan1d(&plan, Rows, HIPFFT_C2C, Cols);
		hipfftSetStream(plan, stream);
		for (int i = 0; i < Bands; i++)
		{
			hipfftExecC2C(plan, Data.FirstAddr() + i * (Rows * Cols), DataTemp.FirstAddr() + i * (Rows * Cols), HIPFFT_BACKWARD);
		}
		hipfftDestroy(plan);

		//hipComplex Value; Value.x = 0.0; Value.y = 0.0;

		StillMatFMul(DataTemp, 1.0 / float(Rows), stream);

		
		// ��Ҫ�ض��ٽض�	
		//if (OriginNum < Rows)
		//{
		//	// ��OriginNum�����ضϣ����У�
		//	Slice(DataTemp, Data, 0, OriginNum-1, 0, Data.dims(1)-1, 0, Data.dims(2)-1, stream);

		//}
		//else
		//{
		//	Data.CopyFromDevice(DataTemp);
		//	OriginNum = Rows;
		//}		

		Slice(DataTemp, Data, 0, OriginNum - 1, 0, Data.dims(1) - 1, 0, Data.dims(2) - 1, stream);

		DataTemp.Free();

	}
	//���������Ϊͬһ������
	// IFFT
	/*IFFT���ٽضϵ�OriginNum ����*/
	void cuIFFT(CCMat &Data, CCMat &Data1, unsigned int OriginNum, int Cols, int Bands,  hipStream_t stream)
	{
		//int Rows = Data.dims(0);
		//int Cols = Data.dims(1);
		//int Bands = Data.dims(2);

		//CCMat DataTemp(Data.dims(0), Data.dims(1), Data.dims(2));

		hipfftHandle plan;
		hipfftPlan1d(&plan, OriginNum, HIPFFT_C2C, Cols);
		hipfftSetStream(plan, stream);
		for (int i = 0; i < Bands; i++)
		{
			hipfftExecC2C(plan, Data.FirstAddr() + i * (OriginNum * Cols), Data1.FirstAddr() + i * (OriginNum * Cols), HIPFFT_BACKWARD);
		}
		hipfftDestroy(plan);

		StillMatFMul(Data1, 1.0 / float(OriginNum), stream);

		Slice(Data1, Data,  0, OriginNum - 1, 0, Data.dims(1) - 1, 0, Data.dims(2) - 1, stream);

		//DataTemp.Free();

	}
	// ������
	void hipConj(CCMat& Data, hipStream_t stream)
	{
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data.elements() + threadsPerBlock - 1) / threadsPerBlock;

		ConjKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (Data.FirstAddr(), Data.elements());
	}

	// ����ѭ���ƶ���shiftNum����0�������ƣ���������
	void cuShiftCol(CCMat& Data, CCMat& Result, int shiftNum, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data.elements() + threadsPerBlock - 1) / threadsPerBlock;
		CirculShiftCol << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data.FirstAddr(), Result.FirstAddr(), shiftNum, Rows, Cols, Bands);
	}

	// ����ѭ���ƶ���shiftNum����0�������ƣ���������
	void cuShiftCol(FCMat& Data, FCMat& Result, int shiftNum, hipStream_t stream)
	{
		int Rows = Data.dims(0);
		int Cols = Data.dims(1);
		int Bands = Data.dims(2);
		int threadsPerBlock = 1024;
		int blocksPerGrid = (Data.elements() + threadsPerBlock - 1) / threadsPerBlock;
		CirculShiftCol << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data.FirstAddr(), Result.FirstAddr(), shiftNum, Rows, Cols, Bands);
	}

	//// ���������
	//double CenterOfMass(Vector<double> envelope) {
	//	//�������������
	//	double Mass = 0;
	//	double area = 0;
	//	double areaLast = 0;
	//	double center = 0;
	//	int flag = 0;
	//	//cout << envelope[0] << " " << envelope[1] << " " << envelope[2] << " " << envelope[3] << " " << envelope[4] << endl;
	//	//envelope[0] = envelope[0] - 0.1;
	//	if (envelope.size() == 1)
	//		center = 0;
	//	else {
	//		Mass = sum(envelope) - envelope[0] / 2 - envelope[envelope.size() - 1] / 2; // �����
	//																					//cout << Mass << endl;
	//		for (int ii = 0; (ii < envelope.size() - 1) && (flag == 0); ii++)
	//		{
	//			area += (envelope[ii] + envelope[ii + 1]) / 2;
	//			areaLast = area - (envelope[ii] + envelope[ii + 1]) / 2;
	//			if (area == Mass / 2) {
	//				//���������envelop[ii+1]��
	//				center = ii + 1;
	//			}
	//			// 			cout << Mass / 2 << endl;
	//			// 			cout << Mass / 2 - envelope[ii] << endl;
	//			if (area > Mass / 2) {
	//				//���������envelop[ii] �� envelop[ii+1]֮��
	//				if (abs(envelope[ii + 1] - envelope[ii]) < 1e-6)
	//				{
	//					center = ii + 0.5;
	//				}
	//				else
	//				{
	//					/*cout << "L2-L1:" << envelope[ii + 1] - envelope[ii] << endl;
	//					cout << "L1^2:"<< pow(envelope[ii], 2) << endl;
	//					cout << "P:" << (Mass / 2 - envelope[ii]) << endl;
	//					cout << "������:" << sqrt(pow(envelope[ii], 2) + 2 * (envelope[ii + 1] - envelope[ii])*(Mass / 2 - envelope[ii])) << endl;*/
	//					center = ii + (sqrt(pow(envelope[ii], 2) + 2 * (envelope[ii + 1] - envelope[ii])*(Mass / 2 - areaLast)) - envelope[ii]) / (envelope[ii + 1] - envelope[ii]);
	//				}
	//				//ii = envelope.size()-1;
	//				flag = 1;
	//			}

	//		}
	//	}
	//	return center;
	//}


}

