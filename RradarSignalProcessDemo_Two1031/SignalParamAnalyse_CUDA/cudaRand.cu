#include "hip/hip_runtime.h"
#include "cudaRand.cuh"

__global__ void setup_kernel(hiprandState *state, unsigned long seed, unsigned int len)
{
	// ��ȡ�̺߳ţ�һά�ṹ
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < len)
	{
		// ��ʼ�������������
		hiprand_init(seed, tid, 0, &state[tid]);
	}

}

__global__ void use(hiprandState *globalState, hipComplex* data, unsigned int len, unsigned int elements)
{
	// ��ȡ�̺߳ţ�һά�ṹ
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < len)
	{
		for (int ii = 0; ii < 128; ii++)
		{
			int idx = ii * len + tid;
			if (idx < elements)
			{
				data[idx] = hiprand_normal2(globalState + tid);
			}
		}
	}
}

__global__ void use(hiprandState *globalState, float* data, unsigned int len, unsigned int elements)
{
	// ��ȡ�̺߳ţ�һά�ṹ
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < len)
	{
		for (int ii = 0; ii < 128; ii++)
		{
			int idx = ii * len + tid;
			if (idx < elements)
			{
				data[idx] = hiprand_normal(globalState + tid);
			}
		}
	}
}

//����(bias, float+bias)�ľ��ȷֲ�
__global__ void cuuseUrand(hiprandState *globalState, float* data, unsigned int len, float scale, float bias, unsigned int elements)
{
	// ��ȡ�̺߳ţ�һά�ṹ
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < len)
	{
		for (int ii = 0; ii < 128; ii++)
		{
			int idx = ii * len + tid;
			if (idx < elements)
			{
				data[idx] = hiprand_uniform(globalState + tid) * scale + bias;
			}
		}
	}
}



//void cuGuassRand(float* GRand, unsigned int N)
//{
//	hiprandGenerator_t gen;
//	// ��������������������������ʽ����ǰΪĬ��
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
//	// �������������
//	hiprandSetPseudoRandomGeneratorSeed(gen, time(0));
//	// ���ɸ�˹����
//	hiprandGenerateNormal(gen, GRand, N, 0.0, 1.0);
//	// ����������
//	hiprandDestroyGenerator(gen);
//}

// ���ɸ�˹�ֲ����������
CCMat Randn(unsigned int Rows, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	CCMat data(Rows, 1, 1);
	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows);
	devStates.Free();

	return data;
}

// ���ɸ�˹�ֲ����������
void Randn(CCMat& data, unsigned int Rows, hipStream_t stream)
{
	data.Resize(Rows, 1, 1);

	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));


	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows);
	devStates.Free();

}

// ���ɸ�˹�ֲ����������
CCMat Randn(unsigned int Rows, unsigned int Cols, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	CCMat data(Rows, Cols, 1);	
	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows*Cols);
	devStates.Free();

	return data;
}

// ���ɸ�˹�ֲ����������
void Randn(CCMat& data, unsigned int Rows, unsigned int Cols, hipStream_t stream)
{
	data.Resize(Rows, Cols, 1);

	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	
	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows*Cols);

	devStates.Free();
}

// ���ɸ�˹�ֲ����������
CCMat Randn(unsigned int Rows, unsigned int Cols, unsigned int Bands, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols*Bands + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	CCMat data(Rows, Cols, Bands);
	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows*Cols*Bands);
	devStates.Free();

	return data;
}

// ���ɸ�˹�ֲ����������
void Randn(CCMat& data, unsigned int Rows, unsigned int Cols, unsigned int Bands, hipStream_t stream)
{
	data.Resize(Rows, Cols, Bands);

	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols*Bands + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	
	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows*Cols*Bands);

	devStates.Free();
}

// ����ʵ����˹�ֲ����������
void Randn(FCMat& data, unsigned int Rows, unsigned int Cols, unsigned int Bands, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols*Bands + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));
	
	data.Resize(Rows, Cols, Bands, 0.0, stream);
	use << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), Rows*Cols*Bands);

	devStates.Free();
}


//����(bias, float+bias)�ľ��ȷֲ�
FCMat Randu(unsigned int Rows, float scale, float bias, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	FCMat data(Rows, 1, 1);
	cuuseUrand << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), scale, bias, Rows);
	devStates.Free();

	return data;
}

//����(bias, float+bias)�ľ��ȷֲ�
void Randu(FCMat& data, unsigned int Rows, float scale, float bias, hipStream_t stream)
{
	data.Resize(Rows, 1, 1);

	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	
	cuuseUrand << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), scale, bias, Rows);

	devStates.Free();
}

//����(bias, float+bias)�ľ��ȷֲ�
FCMat Randu(unsigned int Rows, unsigned int Cols, float scale, float bias, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));
	
	FCMat data(Rows, Cols, 1);
	cuuseUrand << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), scale, bias, Rows*Cols);

	devStates.Free();

	return data;
}

//����(bias, float+bias)�ľ��ȷֲ�
void Randu(FCMat& data, unsigned int Rows, unsigned int Cols, float scale, float bias, hipStream_t stream)
{
	data.Resize(Rows, Cols, 1);
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));
	
	cuuseUrand << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), scale, bias, Rows*Cols);

	devStates.Free();
}

//����(bias, float+bias)�ľ��ȷֲ�
FCMat Randu(unsigned int Rows, unsigned int Cols, unsigned int Bands, float scale, float bias, hipStream_t stream)
{
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols*Bands + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));

	FCMat data(Rows, Cols, Bands);
	cuuseUrand << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), scale, bias, Rows*Cols*Bands);
	devStates.Free();

	return data;
}


//����(bias, float+bias)�ľ��ȷֲ�
void Randu(FCMat& data, unsigned int Rows, unsigned int Cols, unsigned int Bands, float scale, float bias, hipStream_t stream)
{
	data.Resize(Rows, Cols, Bands);
	srand(time(0) * 1e4);
	int threadsPerBlock1 = 128;
	CudaArray<hiprandState> devStates((Rows*Cols*Bands + threadsPerBlock1 - 1) / threadsPerBlock1, 1, 1);
	int blocksPerGrid = (devStates.dims(0) + threadsPerBlock1 - 1) / threadsPerBlock1;
	setup_kernel << <blocksPerGrid, threadsPerBlock1, 0, stream >> >(devStates.FirstAddr(), unsigned long(rand()), devStates.dims(0));
	
	cuuseUrand << <blocksPerGrid, threadsPerBlock1, 0, stream >> > (devStates.FirstAddr(), data.FirstAddr(), devStates.dims(0), scale, bias, Rows*Cols*Bands);

	devStates.Free();
}



