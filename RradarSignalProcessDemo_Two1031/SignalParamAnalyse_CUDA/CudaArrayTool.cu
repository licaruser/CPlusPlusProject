#include "hip/hip_runtime.h"
#include "CudaArrayTool.cuh"

/*��ȡ������ĳ��λ�õ�ֵ*/
template <typename Type>
__global__ void getRowKernel(Type *a, Type *result, int index, int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < elements)
	{
		if (index == idx)
			result = a;
	}
}

/*���þ����е�һ��*/
__global__ void setRowKernel(hipComplex *a, hipComplex *Value, int OneRow, int CurrentBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneRow == R_bias)
			a[CurrentBand * (Row * Col) + C_bias * Row + R_bias] = Value[C_bias];
	}
}
__global__ void setRowKernel(float *a, float *Value, int OneRow, int CurrentBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneRow == R_bias)
			a[CurrentBand * (Row * Col) + C_bias * Row + R_bias] = Value[C_bias];
	}
}
__global__ void setRowKernel(int *a, int *Value, int OneRow, int CurrentBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneRow == R_bias)
			a[CurrentBand * (Row * Col) + C_bias * Row + R_bias] = Value[C_bias];
	}
}

/*���þ����е�һ��*/
__global__ void setColKernel(hipComplex *a, hipComplex *Value, int OneCol, int CurrentBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneCol == C_bias)
			a[CurrentBand * (Row * Col) + C_bias * Row + R_bias] = Value[R_bias];
	}
}
__global__ void setColKernel(float *a, float *Value, int OneCol, int CurrentBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneCol == C_bias)
			a[CurrentBand * (Row * Col) + C_bias * Row + R_bias] = Value[R_bias];
	}
}
__global__ void setColKernel(int *a, int *Value, int OneCol, int CurrentBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneCol == C_bias)
			a[CurrentBand * (Row * Col) + C_bias * Row + R_bias] = Value[R_bias];
	}
}

/*���þ����е�һ����*/
__global__ void setBandKernel(hipComplex *a, hipComplex *Value, int OneBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneBand == B_bias)
			a[B_bias * (Row * Col) + C_bias * Row + R_bias] = Value[C_bias * Row + R_bias];
	}
}
__global__ void setBandKernel(float *a, float *Value, int OneBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneBand == B_bias)
			a[B_bias * (Row * Col) + C_bias * Row + R_bias] = Value[C_bias * Row + R_bias];
	}
}
__global__ void setBandKernel(int *a, int *Value, int OneBand, int Row, int Col, int Band)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < Row * Col * Band)
	{
		unsigned int B_bias = idx / (Row * Col);
		unsigned int C_bias = (idx % (Row * Col)) / Row;
		unsigned int R_bias = (idx % (Row * Col)) % Row;
		if (OneBand == B_bias)
			a[B_bias * (Row * Col) + C_bias * Row + R_bias] = Value[C_bias * Row + R_bias];
	}
}


/*��ֵ�˺���
������*a = elements
*/
__global__ void ValuateKernel(hipComplex *a, hipComplex Value, unsigned int Elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Elements)
	{
		a[idx].x = Value.x;
		a[idx].y = Value.y;
	}
}
__global__ void ValuateKernel(float *a, float Value, unsigned int Elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Elements)
	{
		a[idx] = Value;
	}
}
__global__ void ValuateKernel(int *a, int Value, unsigned int Elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Elements)
	{
		a[idx] = Value;
	}
}
__global__ void ValuateKernel(bool *a, bool Value, unsigned int Elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Elements)
	{
		a[idx] = Value;
	}
}

//*��ȡĳ����
//data Ϊ��������
//Index Ϊ��Ҫ��ȡ����������
//result Ϊ�������
//*/
//template <typename Type>
//void getBands(Type* data, int* Index, Type* result, unsigned int Rows, unsigned int Cols, unsigned int Bands)
//{
//	result = nullptr;
//
//	int threadsPerBlock = 1024;
//	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
//	getBandsKernel << <blocksPerGrid, threadsPerBlock >> >(data, result, OneCol * Rows, Rows * Cols * Bands);
//}


/*����ĳ�У��������µĿռ�*/
void cuSetRow(hipComplex* Data, hipComplex* Value, int CurrentBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, unsigned int OneRow, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setRowKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneRow, CurrentBand, Rows, Cols, Bands);
}
void cuSetRow(float* Data, float* Value, int CurrentBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, unsigned int OneRow, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setRowKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneRow, CurrentBand, Rows, Cols, Bands);
}
void cuSetRow(int* Data, int* Value, int CurrentBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, unsigned int OneRow, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setRowKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneRow, CurrentBand, Rows, Cols, Bands);
}

/*����ĳ�У��������µĿռ�*/
void cuSetCol(hipComplex* Data, hipComplex* Value, int CurrentBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, unsigned int OneCol, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setColKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneCol, CurrentBand, Rows, Cols, Bands);
}
void cuSetCol(float* Data, float* Value, int CurrentBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, unsigned int OneCol, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setColKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneCol, CurrentBand, Rows, Cols, Bands);
}
void cuSetCol(int* Data, int* Value, int CurrentBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, unsigned int OneCol, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setColKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneCol, CurrentBand, Rows, Cols, Bands);
}

/*����ĳ�󣬲������µĿռ�*/
void cuSetBand(hipComplex* Data, hipComplex* Value, int OneBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setBandKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneBand, Rows, Cols, Bands);

}
void cuSetBand(float* Data, float* Value, int OneBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setBandKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneBand, Rows, Cols, Bands);

}
void cuSetBand(int* Data, int* Value, int OneBand, unsigned int Rows, unsigned int Cols, unsigned int Bands, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Rows * Cols * Bands + threadsPerBlock - 1) / threadsPerBlock;
	setBandKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(Data, Value, OneBand, Rows, Cols, Bands);

}


/*��data��ֵΪValue*/
void Valuate(hipComplex* data, hipComplex Value, unsigned int Size, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
	ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data, Value, Size);
}
void Valuate(float* data, float Value, unsigned int Size, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
	ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data, Value, Size);
}
void Valuate(int* data, int Value, unsigned int Size, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
	ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data, Value, Size);
}
void Valuate(bool* data, bool Value, unsigned int Size, hipStream_t stream)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;
	ValuateKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> >(data, Value, Size);
}


//template <typename Type>
//void constants(CudaArray<Type> &data, unsigned int N)
//{
//	data.Resize(N, 1);
//	int threadsPerBlock = 256;
//	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	OneKernel << <blocksPerGrid, threadsPerBlock >> >(data.FirstAddr(), 1, N);
//
//
//}


