#include "hip/hip_runtime.h"

#include"cuDistributeKernel.cuh"

#define PI	double(3.141592653589793)
// ���ɸ�˹�ֲ��Ӳ�����
__global__ void GaussKernel(float sigmaf, float *hf, int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		float f = -6e3 + idx * 12e3 / elements;

		hf[idx] = exp( - f * f / 2 / sigmaf / sigmaf);
	}
}
// ����ָ���ֲ��Ӳ�����
__global__ void ExponentialKernel(float sigmaf, float *hf, int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		float f = -6e3 + idx * 12e3 / elements;

		hf[idx] = exp(-abs(f) / sigmaf);
	}
}

__global__ void CauchyKernel(float sigmaf, float *hf, int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		float f = -6e3 + idx * 12e3 / elements;

		float tmp = abs(f / sigmaf);
		hf[idx] = 1. / (1.0 + pow(tmp, (float)2.0));
	}
}

__global__ void FullSpectrumKernel(float sigmaf, float *hf, int SpectrumPara, int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		float f = -6e3 + idx * 12e3 / elements;

		hf[idx] = 1. / (1.0 + pow(abs(f / sigmaf), (float)SpectrumPara));
	}
}
// ˹ά����� 1��2 ��
__global__ void Swerlling12Kernel(float *Uniformx, float* y, double sigmac, int elements)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < elements)
	{
		y[idx] = -sigmac * log(1.0 - Uniformx[idx]);
	}

}
__global__ void LinearTrans(float *result, float *data, float scale, float bias, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		result[idx] = scale * data[idx] + bias;
	}
}
//��pow
__global__ void cuPow(float *data, float *result, int N, float base)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		result[idx] = std::pow(data[idx], base);
	}
}
//ʵ��+�鲿=����
__global__ void floatToComplex(float *real, float *imag, hipComplex *tar, int len)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < len)
	{
		tar[idx].x = real[idx];
		tar[idx].y = imag[idx];
	}
}
//����=ʵ��+�鲿
__global__ void ComplexTofloat(float *real, float *imag, hipComplex *tar, int len)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < len)
	{
		real[idx] = tar[idx].x;
		imag[idx] = tar[idx].y;
	}
}
// ����ֵ
__global__ void cuAbs(hipComplex *Data, float *Result, int Len)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < Len)
	{
		Result[idx] = hipCabsf(Data[idx]);

	}
}
//������ָ��
__global__ void ComplexIndex(hipComplex *data, int N, float base)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < N)
	{
		//��Ƕ�
		float eps = 10e-10;
		float AngleResult = 0.0;
		if (abs(data[idx].x) <= eps)
			AngleResult = 0.0;
		else if (data[idx].x > 0)
		{
			float angleTmp = data[idx].y / data[idx].x;
			AngleResult = atan(angleTmp);
		}
		else if (data[idx].x <= 0 && data[idx].y >0)
		{
			float angleTmp = data[idx].y / data[idx].x;
			AngleResult = PI + atan(angleTmp);
		}
		else
		{
			float angleTmp = data[idx].y / data[idx].x;
			AngleResult = atan(angleTmp) - PI;
		}
		float Amp = pow(hipCabsf(data[idx]), base);

		//������ָ���������˷�ֵ�����Ǻ�ָ��
		data[idx].x = Amp*cosf(AngleResult * base);
		data[idx].y = Amp*sinf(AngleResult * base);
	}
}

//__global__ void SqrtTileComplexDiv(float *sum, hipComplex *alitemean, hipComplex *result, int N)
//{
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	if (idx < N)
//	{
//		float temp = sum[0] / (N - 1);
//		float sigmma = sqrt(temp);
//		result[idx].x = alitemean[idx].x / sigmma;
//		result[idx].y = alitemean[idx].y / sigmma;
//	}
//}
//__global__ void cuPowATileMean(float *data, float *atilemean, float *sum_mean, float *powatilemean, int N)
//{
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	if (idx < N)
//	{
//		float mean = sum_mean[0] / N;
//		atilemean[idx] = data[idx] - mean;//�ظ�
//		powatilemean[idx] = atilemean[idx] * atilemean[idx];
//	}
//}
//__global__ void cuComplexPowATileMean(hipComplex *data, hipComplex *atilemean, hipComplex *sum_mean, float *powatilemean, int N)
//{
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	if (idx < N)
//	{
//		float mean1 = sum_mean[0].x / N;
//		float mean2 = sum_mean[0].y / N;
//		atilemean[idx].x = data[idx].x - mean1;//�ظ�
//		atilemean[idx].y = data[idx].y - mean2;//�ظ�
//		powatilemean[idx] = atilemean[idx].x * atilemean[idx].x + atilemean[idx].y * atilemean[idx].y;
//	}
//}

/*  float / ��float���� + float�� */
__global__ void MatDivKernel(float *data, float *result, float scale, float bias, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		result[idx] = scale / (data[idx] + bias);
	}
}

//����float���У�start:step:start+(N-1)*step
__global__ void seqKernel(float *result, float start, float step, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		result[idx] = start + step * (float)idx;
	}
}

//�����������Ƶ�һ��������
__global__ void vec2mat(float * data, float *result, int rows, int cols)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int N = rows * cols;
	int bias = idx % rows;
	if (idx < N)
	{
		result[idx] = data[bias];
	}
}

__global__ void Complexvec2mat(hipComplex * data, hipComplex *result, int rows, int cols)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int N = rows * cols;
	int bias = idx % rows;
	if (idx < N)
	{
		result[idx].x = data[bias].x;
		result[idx].y = data[bias].y;
	}
}

//�ɽǶȸ���ŷ����ʽ�õ����ź�
__global__ void EulerFormula(float * sata, hipComplex *result, int N)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
	{
		result[idx].x = cos(sata[idx]);
		result[idx].y = sin(sata[idx]);
	}

}

//����ת��
__global__ void transposeKernel(float* in, float* out, int Rows, int Cols)
{
	long idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < Rows*Cols)
	{
		int R = int(idx / Cols);
		int C = idx % Cols;
		out[C*Rows + R] = in[idx];

	}
}

